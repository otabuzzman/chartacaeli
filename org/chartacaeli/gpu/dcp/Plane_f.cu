#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstdlib>

#include <hip/hip_runtime.h>

#include "Plane_f.h"
#include "Vector3D_f.h"

// from CUDA Toolkit samples
#include <hip/hip_runtime_api.h>

__device__ Plane::Plane( const Vector3D& p1, const Vector3D& p2, const Vector3D& p3 ) {
	set( p1, p2, p3 ) ;
}

__device__ Plane::Plane(
					const float p1x, const float p1y, const float p1z,
					const float p2x, const float p2y, const float p2z,
					const float p3x, const float p3y, const float p3z ) {
	Vector3D p1( p1x, p1y, p1z ) ;
	Vector3D p2( p2x, p2y, p2z ) ;
	Vector3D p3( p3x, p3y, p3z ) ;
	set( p1, p2, p3 ) ;
}

// https://en.wikipedia.org/wiki/Line%E2%80%93plane_intersection#Algebraic_form
__device__ Vector3D& Plane::intersection( const Vector3D& l1, const Vector3D& l2, Vector3D& x ) {
	Vector3D d00( p1 ), l( l2 ), nd0( normal ), ndl( normal ) ;
	float a, b, d ;

	d00.sub( l1 ) ;

	l.sub( l1 ) ;

	a = nd0.dot( d00 ) ;
	b = ndl.dot( l ) ;
	d = __fdividef( a, b ) ;
	l.mul( d ) ;

	x.set( l.x, l.y, l.z ) ;
	x.add( l1 ) ;

	return x ;
}

// private
__device__ void Plane::set( const Vector3D& p1, const Vector3D& p2, const Vector3D& p3 ) {
	Vector3D d21( p2 ) ;
	Vector3D d31( p3 ) ;

	d21.sub( p1 ) ;
	d31.sub( p1 ) ;

	this->p1.set( p1.x, p1.y, p1.z ) ;
	this->p2.set( p2.x, p2.y, p2.z ) ;
	this->p3.set( p3.x, p3.y, p3.z ) ;

	d21.cross( d31 ) ;
	normal.set( d21.x, d21.y, d21.z ) ;
}

#ifdef PLANE_MAIN
// kernel
__global__ void plane( float* buf ) {
	Vector3D p1( 1., 3., 7. ) ;
	Vector3D p2( 3., 7., 1. ) ;
	Vector3D p3( 7., 1., 3. ) ;
	Plane p( p1, p2, p3 ) ;
	Vector3D l0, l1, x ;
	float a, b, c ;
	int i = threadIdx.x ;

	a = i ; b = a+1 ; c = b+1 ;
	l1.set( __fdividef( ( ( a+4 )+( a+1 )+( a-2 ) ), 4.f ), __fdividef( ( ( b+4 )+( b+1 )+( b-2 ) ), 4.f ), __fdividef( ( ( c+4 )+( c+1 )+( c-2 ) ), 4.f ) ) ;
	p.intersection( l0, l1, x ) ;
	buf[3*i] = x.x ;
	buf[3*i+1] = x.y ;
	buf[3*i+2] = x.z ;
}

#define NUM_BLOCKS 1
#define NUM_THREADS 360

int main( int argc, char** argv ) {
	// host buffer
	float buf[3*NUM_THREADS] ;
	// device buffer
	float* dbuf = NULL ;
	hipDeviceProp_t devProp ;
	int devID ;

	// find device and output compute capability on stderr
	devID = gpuGetMaxGflopsDeviceId() ;
	checkCudaErrors( hipSetDevice( devID ) ) ;
	checkCudaErrors( hipGetDeviceProperties( &devProp, devID ) ) ;
	fprintf( stderr, "%d%d\n", devProp.major, devProp.minor ) ;

	// allocate device buffer memory
	checkCudaErrors( hipMalloc( (void**) &dbuf, sizeof( float )*3*NUM_THREADS ) ) ;

	// run kernel
	plane<<<NUM_BLOCKS, NUM_THREADS>>>( dbuf ) ;

	// copy kernel results from device buffer to host
	checkCudaErrors( hipMemcpy( buf, dbuf, sizeof( float )*3*NUM_THREADS, hipMemcpyDeviceToHost ) ) ;
	checkCudaErrors( hipFree( dbuf ) ) ;

	// output result on stdout
	for ( int i=0 ; NUM_THREADS>i ; i++ )
		printf( "%.6f %.6f %.6f\n", buf[3*i], buf[3*i+1], buf[3*i+2] ) ;

	return EXIT_SUCCESS ;
}
#endif // PLANE_MAIN
