#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstdlib>

#include <hip/hip_runtime.h>

#include "Vector3D.h"

// from CUDA Toolkit samples
#include <hip/hip_runtime_api.h>

__device__ Vector3D::Vector3D() : Coordinate( 0, 0, 0 ) {
}

__device__ Vector3D::Vector3D( const Vector3D& v ) : Coordinate( v.x, v.y, v.z ) {
}

__device__ Vector3D::Vector3D( double x, double y, double z ) : Coordinate( x, y, z ) {
}

__device__ Vector3D& Vector3D::add( const Vector3D& v ) {
	x += v.x ;
	y += v.y ;
	z += v.z ;

	return *this ;
}

__device__ Vector3D& Vector3D::sub( const Vector3D& v ) {
	x -= v.x ;
	y -= v.y ;
	z -= v.z ;

	return *this ;
}

__device__ Vector3D& Vector3D::mul( double a ) {
	x *= a ;
	y *= a ;
	z *= a ;

	return *this ;
}

__device__ double Vector3D::dot( const Vector3D& v ) {
	return x*v.x+y*v.y+z*v.z ;
}

__device__ Vector3D& Vector3D::cross( const Vector3D& v ) {
	double x, y, z ;

	x = this->x ;
	y = this->y ;
	z = this->z ;

	this->x = y*v.z-z*v.y ;
	this->y = z*v.x-x*v.z ;
	this->z = x*v.y-y*v.x ;

	return *this ;
}

__device__ Vector3D& Vector3D::apply( const Vector3D& matcol0, const Vector3D& matcol1, const Vector3D& matcol2 ) {
	return apply(
			matcol0.x, matcol1.x, matcol2.x,
			matcol0.y, matcol1.y, matcol2.y,
			matcol0.z, matcol1.z, matcol2.z ) ;
}

__device__ Vector3D& Vector3D::apply(
			const double m00, const double m01, const double m02,
			const double m10, const double m11, const double m12,
			const double m20, const double m21, const double m22 ) {
	double _x = x, _y = y, _z = z ;

	x = _x*m00+_y*m01+_z*m02 ;
	y = _x*m10+_y*m11+_z*m12 ;
	z = _x*m20+_y*m21+_z*m22 ;

	return *this ;
}

#ifdef VECTOR3D_MAIN
// kernel
__global__ void vector3d( double* buf ) {
	Vector3D a, b ;
	int i = threadIdx.x ;

	a.set( i, i+.123, i+.234 ) ;
	b.set( i+.234, i+.123, i ) ;
	buf[i] = a.cross( b ).dot( a ) ;
}

#define NUM_BLOCKS 1
#define NUM_THREADS 360

int main( int argc, char** argv ) {
	// host buffer
	double buf[NUM_THREADS] ;
	// device buffer
	double* dbuf = NULL ;
	hipDeviceProp_t devProp ;
	int devID ;

	// find device and output compute capability on stderr
	devID = gpuGetMaxGflopsDeviceId() ;
	checkCudaErrors( hipSetDevice( devID ) ) ;
	checkCudaErrors( hipGetDeviceProperties( &devProp, devID ) ) ;
	fprintf( stderr, "%d%d\n", devProp.major, devProp.minor ) ;

	// allocate device buffer memory
	checkCudaErrors( hipMalloc( (void**) &dbuf, sizeof( double )*NUM_THREADS ) ) ;

	// run kernel
	vector3d<<<NUM_BLOCKS, NUM_THREADS>>>( dbuf ) ;

	// copy kernel results from device buffer to host
	checkCudaErrors( hipMemcpy( buf, dbuf, sizeof( double )*NUM_THREADS, hipMemcpyDeviceToHost ) ) ;
	checkCudaErrors( hipFree( dbuf ) ) ;

	// output result on stdout
	for ( int i=0 ; NUM_THREADS>i ; i++ )
		printf( "%.6f\n", buf[i] ) ;

	return EXIT_SUCCESS ;
}
#endif // VECTOR3D_MAIN
