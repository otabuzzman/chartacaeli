#include <cstdio>
#include <cstdlib>

#include <hip/hip_runtime.h>

#include "Vector4D.h"

__device__ Vector4D::Vector4D() : e0( 0 ), e1( 0 ), e2( 0 ), e3( 0 ) {
}

__device__ Vector4D::Vector4D( const Vector4D& v ) {
	set( v.e0, v.e1, v.e2, v.e3 ) ;
}

__device__ Vector4D::Vector4D( double e0, double e1, double e2, double e3 ) {
	set( e0, e1, e2, e3 ) ;
}

__device__ void Vector4D::set( double e0, double e1, double e2, double e3 ) {
	this->e0 = e0 ;
	this->e1 = e1 ;
	this->e2 = e2 ;
	this->e3 = e3 ;
}

__device__ Vector4D& Vector4D::apply( const Vector4D& matcol0, const Vector4D& matcol1, const Vector4D& matcol2, const Vector4D& matcol3 ) {
	return apply(
			matcol0.e0, matcol1.e0, matcol2.e0, matcol3.e0,
			matcol0.e1, matcol1.e1, matcol2.e1, matcol3.e1,
			matcol0.e2, matcol1.e2, matcol2.e2, matcol3.e2,
			matcol0.e3, matcol1.e3, matcol2.e3, matcol3.e3 ) ;
}

__device__ Vector4D& Vector4D::apply(
			const double m00, const double m01, const double m02, const double m03,
			const double m10, const double m11, const double m12, const double m13,
			const double m20, const double m21, const double m22, const double m23,
			const double m30, const double m31, const double m32, const double m33 ) {
	double _e0 = e0, _e1 = e1, _e2 = e2, _e3 = e3 ;

	e0 = _e0*m00+_e1*m01+_e2*m02+_e3*m03 ;
	e1 = _e0*m10+_e1*m11+_e2*m12+_e3*m13 ;
	e2 = _e0*m20+_e1*m21+_e2*m22+_e3*m23 ;
	e3 = _e0*m30+_e1*m31+_e2*m32+_e3*m33 ;

	return *this ;
}
