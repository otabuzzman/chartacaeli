#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstdlib>
#include <algorithm>

#include <hip/hip_runtime.h>

#include "RealMatrix.h"

// from CUDA Toolkit samples
#include <hip/hip_runtime_api.h>

__device__ RealMatrix::RealMatrix( double matrix[], int rows, int cols ) {
	set( matrix, rows, cols ) ;
}

__device__ RealMatrix::RealMatrix(
						const double rc00, const double rc01, const double rc02,
						const double rc10, const double rc11, const double rc12,
						const double rc20, const double rc21, const double rc22 ) {
	this->rows = 3 ;
	this->cols = 3 ;
	this->matrix = new double[9] {
			rc00, rc01, rc02,
			rc10, rc11, rc12,
			rc20, rc21, rc22 } ;
}

__device__ RealMatrix::RealMatrix(
						const double rc00, const double rc01, const double rc02, const double rc03,
						const double rc10, const double rc11, const double rc12, const double rc13,
						const double rc20, const double rc21, const double rc22, const double rc23,
						const double rc30, const double rc31, const double rc32, const double rc33 ) {
	this->rows = 4 ;
	this->cols = 4 ;
	this->matrix = new double[16] {
			rc00, rc01, rc02, rc03,
			rc10, rc11, rc12, rc13,
			rc20, rc21, rc22, rc23,
			rc30, rc31, rc32, rc33 } ;
}

__device__ RealMatrix::~RealMatrix() {
	delete[] matrix ;
}

__device__ void RealMatrix::set( double matrix[], int rows, int cols ) {
	int size = rows*cols ;

	this->rows = rows ;
	this->cols = cols ;
	this->matrix = new double[size] ;

	for ( int i=0 ; size>i ; i++ )
		this->matrix[i] = matrix[i] ;
}

__device__ double* RealMatrix::operate( double vector[] ) {
	double sum, *retval = new double[rows] ;

	for ( int r=0 ; rows>r ; r++ ) {
		sum = 0 ;
		for ( int c=0 ; cols>c ; c++ )
			sum += matrix[rows*r+c]*vector[c] ;
		retval[r] = sum ;
	}

	return retval ;
}

#ifdef REALMATRIX_MAIN
// kernel
__global__ void realmatrix( double* buf ) {
	double dat[] = {
		10, 12, 12, 14,
		21, 22, 23, 25,
		32, 32, 34, 36,
		43, 42, 45, 47
	}, vec[4], *res ;
	RealMatrix mat( dat, 4, 4 ) ;
	int i = threadIdx.x ;

	vec[0] = i+.12 ;
	vec[1] = i+1+.34 ;
	vec[2] = i+2+.56 ;
	vec[3] = i+3+.78 ;
	res = mat.operate( vec ) ;
	buf[4*i] = res[0] ;
	buf[4*i+1] = res[1] ;
	buf[4*i+2] = res[2] ;
	buf[4*i+3] = res[3] ;

	delete[] res ;
}

#define NUM_BLOCKS 1
#define NUM_THREADS 360

int main( int argc, char** argv ) {
	// host buffer
	double buf[4*NUM_THREADS] ;
	// device buffer
	double* dbuf = NULL ;
	hipDeviceProp_t devProp ;
	int devID ;

	// find device and output compute capability on stderr
	devID = gpuGetMaxGflopsDeviceId() ;
	checkCudaErrors( hipSetDevice( devID ) ) ;
	checkCudaErrors( hipGetDeviceProperties( &devProp, devID ) ) ;
	fprintf( stderr, "%d%d\n", devProp.major, devProp.minor ) ;

	// allocate device buffer memory
	checkCudaErrors( hipMalloc( (void**) &dbuf, sizeof( double )*4*NUM_THREADS ) ) ;

	// run kernel
	realmatrix<<<NUM_BLOCKS, NUM_THREADS>>>( dbuf ) ;

	// copy kernel results from device buffer to host
	checkCudaErrors( hipMemcpy( buf, dbuf, sizeof( double )*4*NUM_THREADS, hipMemcpyDeviceToHost ) ) ;
	checkCudaErrors( hipFree( dbuf ) ) ;

	// output result on stdout
	for ( int i=0 ; NUM_THREADS>i ; i++ )
		printf( "%.4f %.4f %.4f %.4f\n", buf[4*i], buf[4*i+1], buf[4*i+2], buf[4*i+3] ) ;

	return EXIT_SUCCESS ;
}
#endif // REALMATRIX_MAIN
