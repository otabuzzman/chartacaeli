#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstdlib>
#include <algorithm>

#include <hip/hip_runtime.h>

#include "RealMatrix.h"

// from CUDA Toolkit samples
#include <hip/hip_runtime_api.h>

__device__ RealMatrix::RealMatrix( double matrix[], int rows, int cols ) {
	set( matrix, rows, cols ) ;
}

__device__ RealMatrix::RealMatrix() {}

__device__ RealMatrix::~RealMatrix() {
	delete matrix ;
}

__device__ double* RealMatrix::operate( double vector[] ) {
	double sum, *retval = new double[rows] ;

	for ( int r=0 ; rows>r ; r++ ) {
		sum = 0 ;
		for ( int c=0 ; cols>c ; c++ )
			sum += matrix[rows*r+c]*vector[c] ;
		retval[r] = sum ;
	}

	return retval ;
}

__device__ void RealMatrix::set( double matrix[], int rows, int cols ) {
	int size = rows*cols ;
	this->rows = rows ;
	this->cols = cols ;
	this->matrix = new double[size] ;
	for ( int i=0 ; size>i ; i++ )
		this->matrix[i] = matrix[i] ;
}

#ifdef REALMATRIX_MAIN
// kernel
__global__ void realmatrix( double* buf ) {
	RealMatrix* mat ;
	double dat[] = {
		10, 12, 12, 14,
		21, 22, 23, 25,
		32, 32, 34, 36,
		43, 42, 45, 47
	}, vec[4], *res ;
	int i = threadIdx.x ;

	mat = new RealMatrix( dat, 4, 4 ) ;

	vec[0] = i+.12 ;
	vec[1] = i+1+.34 ;
	vec[2] = i+2+.56 ;
	vec[3] = i+3+.78 ;
	res = mat->operate( vec ) ;
	buf[4*i] = res[0] ;
	buf[4*i+1] = res[1] ;
	buf[4*i+2] = res[2] ;
	buf[4*i+3] = res[3] ;

	delete mat ;
	delete res ;
}

#define NUM_BLOCKS 1
#define NUM_THREADS 360

int main( int argc, char** argv ) {
	// host buffer
	double buf[4*NUM_THREADS] ;
	// device buffer
	double* dbuf = NULL ;
	hipDeviceProp_t devProp ;
	int devID ;

	// find device and output compute capability on stderr
	devID = gpuGetMaxGflopsDeviceId() ;
	checkCudaErrors( hipSetDevice( devID ) ) ;
	checkCudaErrors( hipGetDeviceProperties( &devProp, devID ) ) ;
	fprintf( stderr, "%d%d\n", devProp.major, devProp.minor ) ;

	// allocate device buffer memory
	checkCudaErrors( hipMalloc( (void**) &dbuf, sizeof( double )*4*NUM_THREADS ) ) ;

	// run kernel
	realmatrix<<<NUM_BLOCKS, NUM_THREADS>>>( dbuf ) ;

	// copy kernel results from device buffer to host
	checkCudaErrors( hipMemcpy( buf, dbuf, sizeof( double )*4*NUM_THREADS, hipMemcpyDeviceToHost ) ) ;
	checkCudaErrors( hipFree( dbuf ) ) ;

	// output result on stdout
	for ( int i=0 ; NUM_THREADS>i ; i++ )
		printf( "%.4f %.4f %.4f %.4f\n", buf[4*i], buf[4*i+1], buf[4*i+2], buf[4*i+3] ) ;

	return EXIT_SUCCESS ;
}
#endif // REALMATRIX_MAIN
