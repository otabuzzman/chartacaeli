#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstdlib>

#include <hip/hip_runtime.h>

// from CUDA Toolkit samples
#include <hip/hip_runtime_api.h>

// names inspired by Cg Toolkit Standard Library
__device__ float degrees( float a ) {
	return a*57.295779513082320876798154814105f ;
}
__device__ float radians( float a ) {
	return a*0.017453292519943295769236907685f ;
}

#ifdef MATH_MAIN
// kernel
__global__ void math( float* buf ) {
	buf[threadIdx.x] = degrees( asin ( __sinf( radians( (float) threadIdx.x ) ) ) ) ;
}

#define NUM_BLOCKS 1
#define NUM_THREADS 360

int main( int argc, char** argv ) {
	// host buffer
	float buf[NUM_THREADS] ;
	// device buffer
	float* dbuf = NULL ;
	hipDeviceProp_t devProp ;
	int devID ;

	// find device and output compute capability on stderr
	devID = gpuGetMaxGflopsDeviceId() ;
	checkCudaErrors( hipSetDevice( devID ) ) ;
	checkCudaErrors( hipGetDeviceProperties( &devProp, devID ) ) ;
	fprintf( stderr, "%d%d\n", devProp.major, devProp.minor ) ;

	// allocate device buffer memory
	checkCudaErrors( hipMalloc( (void**) &dbuf, sizeof( float )*NUM_THREADS ) ) ;

	// run kernel
	math<<<NUM_BLOCKS, NUM_THREADS>>>( dbuf ) ;

	// copy kernel results from device buffer to host
	checkCudaErrors( hipMemcpy( buf, dbuf, sizeof( float )*NUM_THREADS, hipMemcpyDeviceToHost ) ) ;
	checkCudaErrors( hipFree( dbuf ) ) ;

	// output result on stdout
	for ( int i=0 ; NUM_THREADS>i ; i++ )
		printf( "%.6f\n", buf[i] ) ;

	return EXIT_SUCCESS ;
}
#endif // MATH_MAIN
