#include <cstdio>
#include <cstdlib>

#include <hip/hip_runtime.h>

// from CUDA Toolkit samples
#include <hip/hip_runtime_api.h>

// names inspired by Cg Toolkit Standard Library
__device__ float degrees( float a ) {
	return a*57.295779513082320876798154814105f ;
}
__device__ float radians( float a ) {
	return a*0.017453292519943295769236907685f ;
}
