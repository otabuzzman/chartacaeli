#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstdlib>

#include <hip/hip_runtime.h>

#include "Coordinate.h"
#include "Math.h"

// from CUDA Toolkit samples
#include <hip/hip_runtime_api.h>

__device__ Coordinate::Coordinate() : x( 0 ), y( 0 ), z( 0 ) {
}

__device__ Coordinate::Coordinate( double c[3] ) {
	set( c[0], c[1], c[2] ) ;
}

__device__ Coordinate::Coordinate( Coordinate& c ) {
	set( c.x, c.y, c.z ) ;
}

__device__ Coordinate::Coordinate( double x, double y, double z ) {
	set( x, y, z ) ;
}

__device__ void Coordinate::set( double x, double y, double z ) {
	this->x = x ;
	this->y = y ;
	this->z = z ;
}

__device__ Coordinate* Coordinate::spherical() {
	double x, y ;
	Coordinate *c ;

	x = degrees( atan2( this->y, this->x ) ) ;
	y = degrees( asin( z/sqrt( this->x*this->x+this->y*this->y+z*z ) ) ) ;

	c = new Coordinate( x, y, 0 ) ;

	return c ;
}

__device__ Coordinate* Coordinate::cartesian() {
	double x, y, z ;
	Coordinate *c ;

	x = radians( cos( this->y ) )*radians( cos( this->x ) ) ;
	y = radians( cos( this->y ) )*radians( sin( this->x ) ) ;
	z = radians( sin( this->y ) ) ;

	c = new Coordinate( x, y, z ) ;

	return c ;
}

__device__ double* Coordinate::toArray() {
	double* r = new double[3] ;

	r[0] = x ;
	r[1] = y ;
	r[2] = z ;

	return r ;
}

// kernel
__global__ void coordinate( double* buf ) {
	Coordinate c( threadIdx.x, threadIdx.x+1, threadIdx.x+2 ), *t0, *t1 ;

	t0 = c.spherical() ;
	t1 = t0->cartesian() ;
	buf[3*threadIdx.x] = t1->x ;
	buf[3*threadIdx.x+1] = t1->y ;
	buf[3*threadIdx.x+2] = t1->z ;
	delete t1 ;
	delete t0 ;
}

#ifdef COORDINATE_MAIN
#define NUM_BLOCKS 1
#define NUM_THREADS 360

int main( int argc, char** argv ) {
	// host buffer
	double buf[3*NUM_THREADS] ;
	// device buffer
	double* dbuf = NULL ;
	hipDeviceProp_t devProp ;
	int devID ;

	// find device and output compute capability on stderr
	devID = gpuGetMaxGflopsDeviceId() ;
	checkCudaErrors( hipSetDevice( devID ) ) ;
	checkCudaErrors( hipGetDeviceProperties( &devProp, devID ) ) ;
	fprintf( stderr, "%d%d\n", devProp.major, devProp.minor ) ;

	// allocate device buffer memory
	checkCudaErrors( hipMalloc( (void**) &dbuf, sizeof( double )*3*NUM_THREADS ) ) ;

	// run kernel
	coordinate<<<NUM_BLOCKS, NUM_THREADS>>>( dbuf ) ;

	// copy kernel results from device buffer to host
	checkCudaErrors( hipMemcpy( buf, dbuf, sizeof( double )*3*NUM_THREADS, hipMemcpyDeviceToHost ) ) ;
	checkCudaErrors( hipFree( dbuf ) ) ;

	// output result on stdout
	for ( int i=0 ; NUM_THREADS>i ; i++ )
		printf( "%.8f %.8f %.8f\n", buf[3*i], buf[3*i+1], buf[3*i+2] ) ;

	return EXIT_SUCCESS ;
}
#endif // COORDINATE_MAIN
