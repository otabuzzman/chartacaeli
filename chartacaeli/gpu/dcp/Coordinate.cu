#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstdlib>

#include <hip/hip_runtime.h>

#include "Coordinate.h"
#include "Math.h"

// from CUDA Toolkit samples
#include <hip/hip_runtime_api.h>

__device__ Coordinate::Coordinate() : x( 0 ), y( 0 ), z( 0 ) {
}

__device__ Coordinate::Coordinate( const Coordinate& c ) {
	set( c.x, c.y, c.z ) ;
}

__device__ Coordinate::Coordinate( double x, double y, double z ) {
	set( x, y, z ) ;
}

__device__ void Coordinate::set( double x, double y, double z ) {
	this->x = x ;
	this->y = y ;
	this->z = z ;
}

__device__ void Coordinate::spherical() {
	double r = sqrt( x*x+y*y+z*z ) ;

	x = degrees( atan2( y, x ) ) ;
	y = degrees( asin( z/r ) ) ;
	z = 0 ;
}

__device__ void Coordinate::cartesian() {
	double _x = x, _y = y ;

	x = cos( radians( _y ) )*cos( radians( _x ) ) ;
	y = cos( radians( _y ) )*sin( radians( _x ) ) ;
	z = sin( radians( _y ) ) ;
}

#ifdef COORDINATE_MAIN
// kernel
__global__ void coordinate( double* buf ) {
	Coordinate c ;

	c.set( threadIdx.x, threadIdx.x+1, threadIdx.x+2 ) ;
	c.spherical() ;
	c.cartesian() ;
	buf[3*threadIdx.x] = c.x ;
	buf[3*threadIdx.x+1] = c.y ;
	buf[3*threadIdx.x+2] = c.z ;
}

#define NUM_BLOCKS 1
#define NUM_THREADS 360

int main( int argc, char** argv ) {
	// host buffer
	double buf[3*NUM_THREADS] ;
	// device buffer
	double* dbuf = NULL ;
	hipDeviceProp_t devProp ;
	int devID ;

	// find device and output compute capability on stderr
	devID = gpuGetMaxGflopsDeviceId() ;
	checkCudaErrors( hipSetDevice( devID ) ) ;
	checkCudaErrors( hipGetDeviceProperties( &devProp, devID ) ) ;
	fprintf( stderr, "%d%d\n", devProp.major, devProp.minor ) ;

	// allocate device buffer memory
	checkCudaErrors( hipMalloc( (void**) &dbuf, sizeof( double )*3*NUM_THREADS ) ) ;

	// run kernel
	coordinate<<<NUM_BLOCKS, NUM_THREADS>>>( dbuf ) ;

	// copy kernel results from device buffer to host
	checkCudaErrors( hipMemcpy( buf, dbuf, sizeof( double )*3*NUM_THREADS, hipMemcpyDeviceToHost ) ) ;
	checkCudaErrors( hipFree( dbuf ) ) ;

	// output result on stdout
	for ( int i=0 ; NUM_THREADS>i ; i++ )
		printf( "%.8f %.8f %.8f\n", buf[3*i], buf[3*i+1], buf[3*i+2] ) ;

	return EXIT_SUCCESS ;
}
#endif // COORDINATE_MAIN
