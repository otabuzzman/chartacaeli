#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstdlib>

#include <hip/hip_runtime.h>

#include "Plane.h"
#include "Vector3D.h"

// from CUDA Toolkit samples
#include <hip/hip_runtime_api.h>

__device__ Plane::Plane( Vector3D& p1, Vector3D& p2, Vector3D& p3 ) {
	set( p1, p2, p3 ) ;
}

__device__ Plane::~Plane() {
	delete p1 ;
	delete p2 ;
	delete p3 ;
	delete normal ;
}

// https://en.wikipedia.org/wiki/Line%E2%80%93plane_intersection#Algebraic_form
__device__ Vector3D* Plane::intersection( Vector3D& l1, Vector3D& l2 ) {
	Vector3D *d00, *l, *x ;
	double d ;

	d00 = new Vector3D( *p1) ;
	d00->sub( l1 ) ;

	l = new Vector3D( l2 ) ;
	l->sub( l1 ) ;

	d = normal->dot( *d00 )/normal->dot( *l ) ;
	l->mul( d ) ;

	x = new Vector3D( l1) ;
	x->add( *l ) ;

	delete d00 ;
	delete l ;

	return x ;
}

__device__ void Plane::set( Vector3D& p1, Vector3D& p2, Vector3D& p3 ) {
	Vector3D* d21 = ( new Vector3D( p2 ) )->sub( p1 ) ;
	Vector3D* d31 = ( new Vector3D( p3 ) )->sub( p1 ) ;

	this->p1 = new Vector3D( p1.x, p1.y, p1.z ) ;
	this->p2 = new Vector3D( p2.x, p2.y, p2.z ) ;
	this->p3 = new Vector3D( p3.x, p3.y, p3.z ) ;

	normal = new Vector3D( *( d21->cross( *d31 ) ) ) ;

	delete d21 ;
	delete d31 ;
}

// kernel
__global__ void plane( double* buf ) {
	Vector3D *p1, *p2, *p3 ;
	Plane* p ;
	Vector3D *l0, *l1, *x ;
	double a, b, c ;
	int i = threadIdx.x ;

	p1 = new Vector3D( 1., 3., 7. ) ;
	p2 = new Vector3D( 3., 7., 1. ) ;
	p3 = new Vector3D( 7., 1., 3. ) ;
	p = new Plane( *p1, *p2, *p3 ) ;

	l0 = new Vector3D() ;
	l1 = new Vector3D() ;

	a = i ; b = a+1 ; c = b+1 ;
	l1->set( ( ( a+4 )+( a+1 )+( a-2 ) )/4, ( ( b+4 )+( b+1 )+( b-2 ) )/4, ( ( c+4 )+( c+1 )+( c-2 ) )/4 ) ;
	x = p->intersection( *l0, *l1 ) ;
	buf[3*i] = x->x ;
	buf[3*i+1] = x->y ;
	buf[3*i+2] = x->z ;

	delete x ;

	delete l1 ;
	delete l0 ;
	delete p ;
	delete p3 ;
	delete p2 ;
	delete p1 ;
}

#ifdef PLANE_MAIN
#define NUM_BLOCKS 1
#define NUM_THREADS 360

int main( int argc, char** argv ) {
	// host buffer
	double buf[3*NUM_THREADS] ;
	// device buffer
	double* dbuf = NULL ;
	hipDeviceProp_t devProp ;
	int devID ;

	// find device and output compute capability on stderr
	devID = gpuGetMaxGflopsDeviceId() ;
	checkCudaErrors( hipSetDevice( devID ) ) ;
	checkCudaErrors( hipGetDeviceProperties( &devProp, devID ) ) ;
	fprintf( stderr, "%d%d\n", devProp.major, devProp.minor ) ;

	// allocate device buffer memory
	checkCudaErrors( hipMalloc( (void**) &dbuf, sizeof( double )*3*NUM_THREADS ) ) ;

	// run kernel
	plane<<<NUM_BLOCKS, NUM_THREADS>>>( dbuf ) ;

	// copy kernel results from device buffer to host
	checkCudaErrors( hipMemcpy( buf, dbuf, sizeof( double )*3*NUM_THREADS, hipMemcpyDeviceToHost ) ) ;
	checkCudaErrors( hipFree( dbuf ) ) ;

	// output result on stdout
	for ( int i=0 ; NUM_THREADS>i ; i++ )
		printf( "%.8f %.8f %.8f\n", buf[3*i], buf[3*i+1], buf[3*i+2] ) ;

	return EXIT_SUCCESS ;
}
#endif // PLANE_MAIN
