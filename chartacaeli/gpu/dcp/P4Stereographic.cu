#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstdlib>

#include <hip/hip_runtime.h>

#include "P4Stereographic.h"
#include "Coordinate.h"
#include "Math.h"

// from CUDA Toolkit samples
#include <hip/hip_runtime_api.h>

__device__ P4Stereographic::P4Stereographic() {
	init( 0, 90, 1, 1 ) ;
}

__device__ void P4Stereographic::init( double lam0, double phi1, double R, double k0 ) {
	this->lam0 = lam0 ;
	this->phi1 = phi1 ;
	sincospi( phi1/180, &sinphi1, &cosphi1 ) ;
	this->R = R ;
	this->k0 = k0 ;

	if ( phi1 == 90 )
		mode = M_NORTH ;
	else if ( phi1 == -90 )
		mode = M_SOUTH ;
	else if ( phi1 == 0 )
		mode = M_EQUATOR ;
	else
		mode = M_OBLIQUE ;
}

__device__ Coordinate& P4Stereographic::forward( const Coordinate& lamphi, Coordinate& xy ) {
	double sinlamdif, coslamdif ;
	double sinphi, cosphi, k, t ;

	sincospi( ( lamphi.x-lam0 )/180, &sinlamdif, &coslamdif ) ;
	sincospi( lamphi.y/180, &sinphi, &cosphi ) ;

	switch ( mode ) {
	case M_NORTH:
		t = tan( radians( 45-lamphi.y/2 ) ) ;
		xy.x = 2*R*k0*t*sinlamdif ;
		xy.y = -2*R*k0*t*coslamdif ;

		break ;
	case M_SOUTH:
		t = tan( radians( 45+lamphi.y/2 ) ) ;
		xy.x = 2*R*k0*t*sinlamdif ;
		xy.y = 2*R*k0*t*coslamdif ;

		break ;
	case M_EQUATOR:
		k = 2*k0/( 1+cosphi*coslamdif ) ;
		xy.x = R*k*cosphi*sinlamdif ;
		xy.y = R*k*sinphi ;

		break ;
	case M_OBLIQUE:
		k = 2*k0/( 1+sinphi1*sinphi+cosphi1*cosphi*coslamdif ) ;
		xy.x = R*k*cosphi*sinlamdif ;
		xy.y = R*k*( cosphi1*sinphi-sinphi1*cosphi*coslamdif ) ;

		break ;
	}

	return xy ;
}

__device__ Coordinate& P4Stereographic::inverse( const Coordinate& xy, Coordinate& lamphi ) {
	double p, c, sinc, cosc ;

	p = sqrt( xy.x*xy.x+xy.y*xy.y ) ;
	c = 2*degrees( atan2( p, 2*R*k0 ) ) ;

	sincospi( c/180, &sinc, &cosc ) ;

	lamphi.y = degrees( asin( cosc*sinphi1+( xy.y*sinc*cosphi1/p ) ) ) ;

	switch ( mode ) {
	case M_NORTH:
		lamphi.x = lam0+degrees( atan2( xy.x, -xy.y ) ) ;

		break ;
	case M_SOUTH:
		lamphi.x = lam0+degrees( atan2( xy.x, xy.y ) ) ;

		break ;
	case M_EQUATOR:
	case M_OBLIQUE:
		lamphi.x = lam0+degrees( atan2( xy.x*sinc, p*cosphi1*cosc-xy.y*sinphi1*sinc ) ) ;

		break ;
	}

	return lamphi ;
}

#ifdef P4STEREOGRAPHIC_MAIN
// kernel
__global__ void p4stereographic( double* buf ) {
	P4Stereographic proj ;
	Coordinate lamphi, xy, res ;
	int i = threadIdx.x ;

	lamphi.set( (double) i, (double) ( i%90 ), 0 ) ;
	proj.forward( lamphi, xy ) ;
	proj.inverse( xy, res ) ;
	buf[2*i] = res.x ;
	buf[2*i+1] = res.y ;
}

#define NUM_BLOCKS 1
#define NUM_THREADS 360

int main( int argc, char** argv ) {
	// host buffer
	double buf[2*NUM_THREADS] ;
	// device buffer
	double* dbuf = NULL ;
	hipDeviceProp_t devProp ;
	int devID ;

	// find device and output compute capability on stderr
	devID = gpuGetMaxGflopsDeviceId() ;
	checkCudaErrors( hipSetDevice( devID ) ) ;
	checkCudaErrors( hipGetDeviceProperties( &devProp, devID ) ) ;
	fprintf( stderr, "%d%d\n", devProp.major, devProp.minor ) ;

	// allocate device buffer memory
	checkCudaErrors( hipMalloc( (void**) &dbuf, sizeof( double )*2*NUM_THREADS ) ) ;

	// run kernel
	p4stereographic<<<NUM_BLOCKS, NUM_THREADS>>>( dbuf ) ;

	// copy kernel results from device buffer to host
	checkCudaErrors( hipMemcpy( buf, dbuf, sizeof( double )*2*NUM_THREADS, hipMemcpyDeviceToHost ) ) ;
	checkCudaErrors( hipFree( dbuf ) ) ;

	// output result on stdout
	for ( int i=0 ; NUM_THREADS>i ; i++ )
		printf( "%.4f %.4f\n", buf[2*i], buf[2*i+1] ) ;

	return EXIT_SUCCESS ;
}
#endif // P4STEREOGRAPHIC_MAIN
