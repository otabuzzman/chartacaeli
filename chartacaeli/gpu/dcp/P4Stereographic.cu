#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstdlib>

#include <hip/hip_runtime.h>

#include "P4Stereographic.h"
#include "Coordinate.h"
#include "Math.h"

// from CUDA Toolkit samples
#include <hip/hip_runtime_api.h>

__device__ P4Stereographic::P4Stereographic() {
	init( 0, 90, 1, 1 ) ;
}

__device__ void P4Stereographic::init( double lam0, double phi1, double R, double k0 ) {
	this->lam0 = lam0 ;
	this->phi1 = phi1 ;
	sinphi1 = sin( radians( phi1 ) ) ;
	cosphi1 = cos( radians( phi1 ) ) ;
	this->R = R ;
	this->k0 = k0 ;

	if ( phi1 == 90 )
		mode = M_NORTH ;
	else if ( phi1 == -90 )
		mode = M_SOUTH ;
	else if ( phi1 == 0 )
		mode = M_EQUATOR ;
	else
		mode = M_OBLIQUE ;
}

__device__ Coordinate* P4Stereographic::forward( Coordinate& lamphi ) {
	Coordinate* xy = new Coordinate() ;
	double sinlamdif, coslamdif ;
	double sinphi, cosphi, k, t ;

	sinlamdif = sin( radians( lamphi.x-lam0 ) ) ;
	coslamdif = cos( radians( lamphi.x-lam0 ) ) ;
	sinphi = sin( radians( lamphi.y ) ) ;
	cosphi = cos( radians( lamphi.y ) ) ;

	switch ( mode ) {
	case M_NORTH:
		t = tan( radians( 45-lamphi.y/2 ) ) ;
		xy->x = 2*R*k0*t*sinlamdif ;
		xy->y = -2*R*k0*t*coslamdif ;

		break ;
	case M_SOUTH:
		t = tan( radians( 45+lamphi.y/2 ) ) ;
		xy->x = 2*R*k0*t*sinlamdif ;
		xy->y = 2*R*k0*t*coslamdif ;

		break ;
	case M_EQUATOR:
		k = 2*k0/( 1+cosphi*coslamdif ) ;
		xy->x = R*k*cosphi*sinlamdif ;
		xy->y = R*k*sinphi ;

		break ;
	case M_OBLIQUE:
		k = 2*k0/( 1+sinphi1*sinphi+cosphi1*cosphi*coslamdif ) ;
		xy->x = R*k*cosphi*sinlamdif ;
		xy->y = R*k*( cosphi1*sinphi-sinphi1*cosphi*coslamdif ) ;

		break ;
	}

	return xy ;
}

__device__ Coordinate* P4Stereographic::inverse( Coordinate& xy ) {
	Coordinate* lamphi = new Coordinate() ;
	double p, c, sinc, cosc ;

	p = pow( xy.x*xy.x+xy.y*xy.y, .5 ) ;
	c = 2*degrees( atan2( p, 2*R*k0 ) ) ;

	sinc = sin( radians( c ) ) ;
	cosc = cos( radians( c ) ) ;

	lamphi->y = degrees( asin( cosc*sinphi1+( xy.y*sinc*cosphi1/p ) ) ) ;

	switch ( mode ) {
	case M_NORTH:
		lamphi->x = lam0+degrees( atan2( xy.x, -xy.y ) ) ;

		break ;
	case M_SOUTH:
		lamphi->x = lam0+degrees( atan2( xy.x, xy.y ) ) ;

		break ;
	case M_EQUATOR:
	case M_OBLIQUE:
		lamphi->x = lam0+degrees( atan2( xy.x*sinc, p*cosphi1*cosc-xy.y*sinphi1*sinc ) ) ;

		break ;
	}

	return lamphi ;
}

// kernel
__global__ void p4stereographic( double* buf ) {
	P4Projector* proj ;
	Coordinate *lamphi, *xy, *res ;
	int i = threadIdx.x ;

	proj = new P4Stereographic() ;
	lamphi = new Coordinate() ;

	lamphi->set( (double) i, (double) ( i%90 ), 0 ) ;
	xy = proj->forward( *lamphi ) ;
	res = proj->inverse( *xy ) ;
	buf[2*i] = res->x ;
	buf[2*i+1] = res->y ;

	delete proj ;
	delete lamphi ;
	delete xy ;
	delete res ;
}

#ifdef P4STEREOGRAPHIC_MAIN
#define NUM_BLOCKS 1
#define NUM_THREADS 360

int main( int argc, char** argv ) {
	// host buffer
	double buf[2*NUM_THREADS] ;
	// device buffer
	double* dbuf = NULL ;
	hipDeviceProp_t devProp ;
	int devID ;

	// find device and output compute capability on stderr
	devID = gpuGetMaxGflopsDeviceId() ;
	checkCudaErrors( hipSetDevice( devID ) ) ;
	checkCudaErrors( hipGetDeviceProperties( &devProp, devID ) ) ;
	fprintf( stderr, "%d%d\n", devProp.major, devProp.minor ) ;

	// allocate device buffer memory
	checkCudaErrors( hipMalloc( (void**) &dbuf, sizeof( double )*2*NUM_THREADS ) ) ;

	// run kernel
	p4stereographic<<<NUM_BLOCKS, NUM_THREADS>>>( dbuf ) ;

	// copy kernel results from device buffer to host
	checkCudaErrors( hipMemcpy( buf, dbuf, sizeof( double )*2*NUM_THREADS, hipMemcpyDeviceToHost ) ) ;
	checkCudaErrors( hipFree( dbuf ) ) ;

	// output result on stdout
	for ( int i=0 ; NUM_THREADS>i ; i++ )
		printf( "%.4f %.4f\n", buf[2*i], buf[2*i+1] ) ;

	return EXIT_SUCCESS ;
}
#endif // P4STEREOGRAPHIC_MAIN
