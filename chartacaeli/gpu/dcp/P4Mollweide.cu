#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstdlib>

#include <hip/hip_math_constants.h>

#include "P4Mollweide.h"
#include "Coordinate.h"
#include "Math.h"

// from CUDA Toolkit samples
#include <hip/hip_runtime_api.h>

__device__ P4Mollweide::P4Mollweide() : V_CON( 1e-7 ),
								radperdeg( 3.141592653589793/180. ) ,
								degperrad( 180./3.141592653589793 ) {
	init( 0, 0, 1, 1 ) ;
}

__device__ void P4Mollweide::init( double lam0, double phi1, double R, double k0 ) {
	this->lam0 = lam0 ;
	this->R = R ;
}

__device__ Coordinate* P4Mollweide::forward( Coordinate& lamphi ) {
	Coordinate* xy = new Coordinate() ;
	double tht2 = lamphi.y, dtht2 = 0, sintht2, costht2 ;
	double sinphi, tht, sintht, costht ;

	sinphi = sin( radians( lamphi.y ) ) ;

	do {
		tht2 = tht2+dtht2 ;

		sintht2 = sin( radians( tht2 ) ) ;
		costht2 = cos( radians( tht2 ) ) ;

		dtht2 = -( tht2*radperdeg+sintht2-HIP_PI*sinphi )/( 1+costht2 )*degperrad ;
	} while ( abs( dtht2 )>V_CON ) ;

	tht = tht2*.5 ;
	sintht = sin( radians( tht ) ) ;
	costht = cos( radians( tht ) ) ;

	xy->x = ( pow( 8., .5 )/HIP_PI )*R*( lamphi.x-lam0 )*costht*radperdeg ;
	xy->y = pow( 2., .5 )*R*sintht ;

	return xy ;
}

__device__ Coordinate* P4Mollweide::inverse( Coordinate& xy ) {
	Coordinate* lamphi = new Coordinate() ;
	double tht, sin2tht, costht ;

	tht = degrees( asin( xy.y/( pow( 2., .5 )*R ) ) ) ;

	sin2tht = sin( radians( 2*tht ) ) ;
	lamphi->y = degrees( asin( ( 2*tht*radperdeg+sin2tht )/HIP_PI ) ) ;

	if ( abs( lamphi->y ) == 90 )
		lamphi->x = lam0 ;
	else {
		costht = cos( radians( tht ) ) ;
		lamphi->x = lam0+( HIP_PI*xy.x/( pow( 8., .5 )*R*costht ) )*degperrad ;
	}

	return lamphi ;
}

#ifdef P4MOLLWEIDE_MAIN
// kernel
__global__ void p4mollweide( double* buf ) {
	P4Projector* proj ;
	Coordinate *lamphi, *xy, *res ;
	int i = threadIdx.x ;

	proj = new P4Mollweide() ;
	lamphi = new Coordinate() ;

	lamphi->set( (double) i, (double) ( i%90 ), 0 ) ;
	xy = proj->forward( *lamphi ) ;
	res = proj->inverse( *xy ) ;
	buf[2*i] = res->x ;
	buf[2*i+1] = res->y ;

	delete proj ;
	delete lamphi ;
	delete xy ;
	delete res ;
}

#define NUM_BLOCKS 1
#define NUM_THREADS 360

int main( int argc, char** argv ) {
	// host buffer
	double buf[2*NUM_THREADS] ;
	// device buffer
	double* dbuf = NULL ;
	hipDeviceProp_t devProp ;
	int devID ;

	// find device and output compute capability on stderr
	devID = gpuGetMaxGflopsDeviceId() ;
	checkCudaErrors( hipSetDevice( devID ) ) ;
	checkCudaErrors( hipGetDeviceProperties( &devProp, devID ) ) ;
	fprintf( stderr, "%d%d\n", devProp.major, devProp.minor ) ;

	// allocate device buffer memory
	checkCudaErrors( hipMalloc( (void**) &dbuf, sizeof( double )*2*NUM_THREADS ) ) ;

	// run kernel
	p4mollweide<<<NUM_BLOCKS, NUM_THREADS>>>( dbuf ) ;

	// copy kernel results from device buffer to host
	checkCudaErrors( hipMemcpy( buf, dbuf, sizeof( double )*2*NUM_THREADS, hipMemcpyDeviceToHost ) ) ;
	checkCudaErrors( hipFree( dbuf ) ) ;

	// output result on stdout
	for ( int i=0 ; NUM_THREADS>i ; i++ )
		printf( "%.4f %.4f\n", buf[2*i], buf[2*i+1] ) ;

	return EXIT_SUCCESS ;
}
#endif // P4MOLLWEIDE_MAIN
