#include <cstdio>
#include <cstdlib>

#include <hip/hip_runtime.h>

#include "Vector4D.h"

__device__ Vector4D::Vector4D() : e0( 0 ), e1( 0 ), e2( 0 ), e3( 0 ) {
}

__device__ Vector4D::Vector4D( const Vector4D& v ) {
	set( v.e0, v.e1, v.e2, v.e3 ) ;
}

__device__ Vector4D::Vector4D( double e0, double e1, double e2, double e3 ) {
	set( e0, e1, e2, e3 ) ;
}

__device__ void Vector4D::set( double e0, double e1, double e2, double e3 ) {
	this->e0 = e0 ;
	this->e1 = e1 ;
	this->e2 = e2 ;
	this->e3 = e3 ;
}

__device__ Vector4D& Vector4D::apply( const Vector4D& matcol0, const Vector4D& matcol1, const Vector4D& matcol2, const Vector4D& matcol3 ) {
	return apply(
			matcol0.e0, matcol1.e0, matcol2.e0, matcol3.e0,
			matcol0.e1, matcol1.e1, matcol2.e1, matcol3.e1,
			matcol0.e2, matcol1.e2, matcol2.e2, matcol3.e2,
			matcol0.e3, matcol1.e3, matcol2.e3, matcol3.e3 ) ;
}

__device__ Vector4D& Vector4D::apply(
			const double m00, const double m01, const double m02, const double m03,
			const double m10, const double m11, const double m12, const double m13,
			const double m20, const double m21, const double m22, const double m23,
			const double m30, const double m31, const double m32, const double m33 ) {
	e0 = e0*m00+e0*m01+e0*m02+e0*m03 ;
	e1 = e1*m10+e1*m11+e1*m12+e1*m13 ;
	e2 = e2*m20+e2*m21+e2*m22+e2*m23 ;
	e3 = e3*m30+e3*m31+e3*m32+e3*m33 ;

	return *this ;
}
