#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstdlib>

#include <hip/hip_runtime.h>

#include "Plane_f.h"
#include "Vector3D_f.h"

// from CUDA Toolkit samples
#include <hip/hip_runtime_api.h>

__device__ Plane::Plane( const Vector3D& p1, const Vector3D& p2, const Vector3D& p3 ) {
	set( p1, p2, p3 ) ;
}

__device__ Plane::Plane(
					const float p1x, const float p1y, const float p1z,
					const float p2x, const float p2y, const float p2z,
					const float p3x, const float p3y, const float p3z ) {
	Vector3D p1( p1x, p1y, p1z ) ;
	Vector3D p2( p2x, p2y, p2z ) ;
	Vector3D p3( p3x, p3y, p3z ) ;
	set( p1, p2, p3 ) ;
}

// https://en.wikipedia.org/wiki/Line%E2%80%93plane_intersection#Algebraic_form
__device__ Vector3D& Plane::intersection( const Vector3D& l1, const Vector3D& l2, Vector3D& x ) {
	Vector3D d00( p1 ), l( l2 ), nd0( normal ), ndl( normal ) ;
	float a, b, d ;

	d00.sub( l1 ) ;

	l.sub( l1 ) ;

	a = nd0.dot( d00 ) ;
	b = ndl.dot( l ) ;
	d = __fdividef( a, b ) ;
	l.mul( d ) ;

	x.set( l.x, l.y, l.z ) ;
	x.add( l1 ) ;

	return x ;
}

// private
__device__ void Plane::set( const Vector3D& p1, const Vector3D& p2, const Vector3D& p3 ) {
	Vector3D d21( p2 ) ;
	Vector3D d31( p3 ) ;

	d21.sub( p1 ) ;
	d31.sub( p1 ) ;

	this->p1.set( p1.x, p1.y, p1.z ) ;
	this->p2.set( p2.x, p2.y, p2.z ) ;
	this->p3.set( p3.x, p3.y, p3.z ) ;

	d21.cross( d31 ) ;
	normal.set( d21.x, d21.y, d21.z ) ;
}
