#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstdlib>

#include "P4Orthographic.h"
#include "Coordinate.h"
#include "Math.h"

// from CUDA Toolkit samples
#include <hip/hip_runtime_api.h>

__device__ P4Orthographic::P4Orthographic() {
	init( 0, 90, 1, 1 ) ;
}

__device__ void P4Orthographic::init( double lam0, double phi1, double R, double k0 ) {
	this->lam0 = lam0 ;
	this->phi1 = phi1 ;
	sincospi( phi1/180, &sinphi1, &cosphi1 ) ;
	this->R = R ;

	if ( phi1 == 90 )
		mode = M_NORTH ;
	else if ( phi1 == -90 )
		mode = M_SOUTH ;
	else if ( phi1 == 0 )
		mode = M_EQUATOR ;
	else
		mode = M_OBLIQUE ;
}

__device__ Coordinate& P4Orthographic::forward( const Coordinate& lamphi, Coordinate& xy ) {
	double sinlamdif, coslamdif ;
	double sinphi, cosphi ;

	sincospi( ( lamphi.x-lam0 )/180, &sinlamdif, &coslamdif ) ;
	sincospi( lamphi.y/180, &sinphi, &cosphi ) ;

	xy.x = R*cosphi*sinlamdif ;

	switch ( mode ) {
	case M_NORTH:
		xy.y = -R*cosphi*coslamdif ;

		break ;
	case M_SOUTH:
		xy.y = R*cosphi*coslamdif ;

		break ;
	case M_EQUATOR:
		xy.y = R*sinphi ;

		break ;
	case M_OBLIQUE:
		xy.y = R*( cosphi1*sinphi-sinphi1*cosphi*coslamdif ) ;

		break ;
	}

	return xy ;
}

__device__ Coordinate& P4Orthographic::inverse( const Coordinate& xy, Coordinate& lamphi ) {
	double p, c, sinc, cosc ;

	p = sqrt( xy.x*xy.x+xy.y*xy.y ) ;
	c = degrees( asin( p/R ) ) ;

	sincospi( c/180, &sinc, &cosc ) ;

	lamphi.y = degrees( asin( cosc*sinphi1+( xy.y*sinc*cosphi1/p ) ) ) ;

	switch ( mode ) {
	case M_NORTH:
		lamphi.x = lam0+degrees( atan2(xy.x, -xy.y ) ) ;

		break ;
	case M_SOUTH:
		lamphi.x = lam0+degrees( atan2(xy.x, xy.y ) ) ;

		break ;
	case M_EQUATOR:
	case M_OBLIQUE:
		lamphi.x = lam0+degrees( atan2( xy.x*sinc, p*cosphi1*cosc-xy.y*sinphi1*sinc ) ) ;

		break ;
	}

	return lamphi ;
}

#ifdef P4ORTHOGRAPHIC_MAIN
// kernel
__global__ void p4orthographic( double* buf ) {
	P4Orthographic proj ;
	Coordinate lamphi, xy, res ;
	int i = threadIdx.x ;

	lamphi.set( (double) i, (double) ( i%90 ), 0 ) ;
	proj.forward( lamphi, xy ) ;
	proj.inverse( xy, res ) ;
	buf[2*i] = res.x ;
	buf[2*i+1] = res.y ;
}

#define NUM_BLOCKS 1
#define NUM_THREADS 360

int main( int argc, char** argv ) {
	// host buffer
	double buf[2*NUM_THREADS] ;
	// device buffer
	double* dbuf = NULL ;
	hipDeviceProp_t devProp ;
	int devID ;

	// find device and output compute capability on stderr
	devID = gpuGetMaxGflopsDeviceId() ;
	checkCudaErrors( hipSetDevice( devID ) ) ;
	checkCudaErrors( hipGetDeviceProperties( &devProp, devID ) ) ;
	fprintf( stderr, "%d%d\n", devProp.major, devProp.minor ) ;

	// allocate device buffer memory
	checkCudaErrors( hipMalloc( (void**) &dbuf, sizeof( double )*2*NUM_THREADS ) ) ;

	// run kernel
	p4orthographic<<<NUM_BLOCKS, NUM_THREADS>>>( dbuf ) ;

	// copy kernel results from device buffer to host
	checkCudaErrors( hipMemcpy( buf, dbuf, sizeof( double )*2*NUM_THREADS, hipMemcpyDeviceToHost ) ) ;
	checkCudaErrors( hipFree( dbuf ) ) ;

	// output result on stdout
	for ( int i=0 ; NUM_THREADS>i ; i++ )
		printf( "%.4f %.4f\n", buf[2*i], buf[2*i+1] ) ;

	return EXIT_SUCCESS ;
}
#endif // P4ORTHOGRAPHIC_MAIN
