#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstdlib>

#include "P4Orthographic.h"
#include "Coordinate.h"
#include "Math.h"

// from CUDA Toolkit samples
#include <hip/hip_runtime_api.h>

__device__ P4Orthographic::P4Orthographic() {
	init( 0, 90, 1, 1 ) ;
}

__device__ void P4Orthographic::init( double lam0, double phi1, double R, double k0 ) {
	this->lam0 = lam0 ;
	this->phi1 = phi1 ;
	sinphi1 = sin( radians( phi1 ) ) ;
	cosphi1 = cos( radians( phi1 ) ) ;
	this->R = R ;

	if ( phi1 == 90 )
		mode = M_NORTH ;
	else if ( phi1 == -90 )
		mode = M_SOUTH ;
	else if ( phi1 == 0 )
		mode = M_EQUATOR ;
	else
		mode = M_OBLIQUE ;
}

__device__ Coordinate* P4Orthographic::forward( Coordinate& lamphi ) {
	Coordinate* xy = new Coordinate() ;
	double sinlamdif, coslamdif ;
	double sinphi, cosphi ;

	sinlamdif = sin( radians( lamphi.x-lam0 ) ) ;
	coslamdif = cos( radians( lamphi.x-lam0 ) ) ;
	sinphi = sin( radians( lamphi.y ) ) ;
	cosphi = cos( radians( lamphi.y ) ) ;

	xy->x = R*cosphi*sinlamdif ;

	switch ( mode ) {
	case M_NORTH:
		xy->y = -R*cosphi*coslamdif ;

		break ;
	case M_SOUTH:
		xy->y = R*cosphi*coslamdif ;

		break ;
	case M_EQUATOR:
		xy->y = R*sinphi ;

		break ;
	case M_OBLIQUE:
		xy->y = R*( cosphi1*sinphi-sinphi1*cosphi*coslamdif ) ;

		break ;
	}

	return xy ;
}

__device__ Coordinate* P4Orthographic::inverse( Coordinate& xy ) {
	Coordinate* lamphi = new Coordinate() ;
	double p, c, sinc, cosc ;

	p = pow( xy.x*xy.x+xy.y*xy.y, .5 ) ;
	c = degrees( asin( p/R ) ) ;

	sinc = sin( radians( c ) ) ;
	cosc = cos( radians( c ) ) ;

	lamphi->y = degrees( asin( cosc*sinphi1+( xy.y*sinc*cosphi1/p ) ) ) ;

	switch ( mode ) {
	case M_NORTH:
		lamphi->x = lam0+degrees( atan2(xy.x, -xy.y ) ) ;

		break ;
	case M_SOUTH:
		lamphi->x = lam0+degrees( atan2(xy.x, xy.y ) ) ;

		break ;
	case M_EQUATOR:
	case M_OBLIQUE:
		lamphi->x = lam0+degrees( atan2( xy.x*sinc, p*cosphi1*cosc-xy.y*sinphi1*sinc ) ) ;

		break ;
	}

	return lamphi ;
}

// kernel
__global__ void p4orthographic( double* buf ) {
	P4Projector* proj ;
	Coordinate *lamphi, *xy, *res ;
	int i = threadIdx.x ;

	proj = new P4Orthographic() ;
	lamphi = new Coordinate() ;

	lamphi->set( (double) i, (double) ( i%90 ), 0 ) ;
	xy = proj->forward( *lamphi ) ;
	res = proj->inverse( *xy ) ;
	buf[2*i] = res->x ;
	buf[2*i+1] = res->y ;

	delete proj ;
	delete lamphi ;
	delete xy ;
	delete res ;
}

#ifdef P4ORTHOGRAPHIC_MAIN
#define NUM_BLOCKS 1
#define NUM_THREADS 360

int main( int argc, char** argv ) {
	// host buffer
	double buf[2*NUM_THREADS] ;
	// device buffer
	double* dbuf = NULL ;
	hipDeviceProp_t devProp ;
	int devID ;

	// find device and output compute capability on stderr
	devID = gpuGetMaxGflopsDeviceId() ;
	checkCudaErrors( hipSetDevice( devID ) ) ;
	checkCudaErrors( hipGetDeviceProperties( &devProp, devID ) ) ;
	fprintf( stderr, "%d%d\n", devProp.major, devProp.minor ) ;

	// allocate device buffer memory
	checkCudaErrors( hipMalloc( (void**) &dbuf, sizeof( double )*2*NUM_THREADS ) ) ;

	// run kernel
	p4orthographic<<<NUM_BLOCKS, NUM_THREADS>>>( dbuf ) ;

	// copy kernel results from device buffer to host
	checkCudaErrors( hipMemcpy( buf, dbuf, sizeof( double )*2*NUM_THREADS, hipMemcpyDeviceToHost ) ) ;
	checkCudaErrors( hipFree( dbuf ) ) ;

	// output result on stdout
	for ( int i=0 ; NUM_THREADS>i ; i++ )
		printf( "%.8f %.8f\n", buf[2*i], buf[2*i+1] ) ;

	return EXIT_SUCCESS ;
}
#endif // P4ORTHOGRAPHIC_MAIN
