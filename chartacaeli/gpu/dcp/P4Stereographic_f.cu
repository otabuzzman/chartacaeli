#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstdlib>

#include <hip/hip_runtime.h>

#include "P4Stereographic_f.h"
#include "Coordinate_f.h"
#include "Math_f.h"

// from CUDA Toolkit samples
#include <hip/hip_runtime_api.h>

__device__ P4Stereographic::P4Stereographic() {
	init( 0, 90, 1, 1 ) ;
}

__device__ void P4Stereographic::init( float lam0, float phi1, float R, float k0 ) {
	this->lam0 = lam0 ;
	this->phi1 = phi1 ;
	sincospif( __fdividef( phi1, 180.f ), &sinphi1, &cosphi1 ) ;
	this->R = R ;
	this->k0 = k0 ;

	if ( phi1 == 90 )
		mode = M_NORTH ;
	else if ( phi1 == -90 )
		mode = M_SOUTH ;
	else if ( phi1 == 0 )
		mode = M_EQUATOR ;
	else
		mode = M_OBLIQUE ;
}

__device__ Coordinate& P4Stereographic::forward( const Coordinate& lamphi, Coordinate& xy ) {
	float sinlamdif, coslamdif ;
	float sinphi, cosphi, k, t ;

	sincospif( __fdividef( ( lamphi.x-lam0 ), 180.f ), &sinlamdif, &coslamdif ) ;
	sincospif( __fdividef( lamphi.y, 180.f ), &sinphi, &cosphi ) ;

	switch ( mode ) {
	case M_NORTH:
		t = __tanf( radians( 45-__fdividef( lamphi.y, 2.f ) ) ) ;
		xy.x = 2*R*k0*t*sinlamdif ;
		xy.y = -2*R*k0*t*coslamdif ;

		break ;
	case M_SOUTH:
		t = __tanf( radians( 45+__fdividef( lamphi.y, 2.f ) ) ) ;
		xy.x = 2*R*k0*t*sinlamdif ;
		xy.y = 2*R*k0*t*coslamdif ;

		break ;
	case M_EQUATOR:
		k = 2*__fdividef( k0, ( 1+cosphi*coslamdif ) ) ;
		xy.x = R*k*cosphi*sinlamdif ;
		xy.y = R*k*sinphi ;

		break ;
	case M_OBLIQUE:
		k = 2*__fdividef( k0, ( 1+sinphi1*sinphi+cosphi1*cosphi*coslamdif ) ) ;
		xy.x = R*k*cosphi*sinlamdif ;
		xy.y = R*k*( cosphi1*sinphi-sinphi1*cosphi*coslamdif ) ;

		break ;
	}

	return xy ;
}

__device__ Coordinate& P4Stereographic::inverse( const Coordinate& xy, Coordinate& lamphi ) {
	float p, c, sinc, cosc ;

	p = sqrtf( xy.x*xy.x+xy.y*xy.y ) ;
	c = 2*degrees( atan2f( p, 2*R*k0 ) ) ;

	sincospif( __fdividef( c, 180.f ), &sinc, &cosc ) ;

	lamphi.y = degrees( asinf( cosc*sinphi1+( xy.y*sinc*__fdividef( cosphi1, p ) ) ) ) ;

	switch ( mode ) {
	case M_NORTH:
		lamphi.x = lam0+degrees( atan2f( xy.x, -xy.y ) ) ;

		break ;
	case M_SOUTH:
		lamphi.x = lam0+degrees( atan2f( xy.x, xy.y ) ) ;

		break ;
	case M_EQUATOR:
	case M_OBLIQUE:
		lamphi.x = lam0+degrees( atan2f( xy.x*sinc, p*cosphi1*cosc-xy.y*sinphi1*sinc ) ) ;

		break ;
	}

	return lamphi ;
}

#ifdef P4STEREOGRAPHIC_MAIN
// kernel
__global__ void p4stereographic( float* buf ) {
	P4Stereographic proj ;
	Coordinate lamphi, xy, res ;
	int i = threadIdx.x ;

	lamphi.set( (float) i, (float) ( i%90 ), 0 ) ;
	proj.forward( lamphi, xy ) ;
	proj.inverse( xy, res ) ;
	buf[2*i] = res.x ;
	buf[2*i+1] = res.y ;
}

#define NUM_BLOCKS 1
#define NUM_THREADS 360

int main( int argc, char** argv ) {
	// host buffer
	float buf[2*NUM_THREADS] ;
	// device buffer
	float* dbuf = NULL ;
	hipDeviceProp_t devProp ;
	int devID ;

	// find device and output compute capability on stderr
	devID = gpuGetMaxGflopsDeviceId() ;
	checkCudaErrors( hipSetDevice( devID ) ) ;
	checkCudaErrors( hipGetDeviceProperties( &devProp, devID ) ) ;
	fprintf( stderr, "%d%d\n", devProp.major, devProp.minor ) ;

	// allocate device buffer memory
	checkCudaErrors( hipMalloc( (void**) &dbuf, sizeof( float )*2*NUM_THREADS ) ) ;

	// run kernel
	p4stereographic<<<NUM_BLOCKS, NUM_THREADS>>>( dbuf ) ;

	// copy kernel results from device buffer to host
	checkCudaErrors( hipMemcpy( buf, dbuf, sizeof( float )*2*NUM_THREADS, hipMemcpyDeviceToHost ) ) ;
	checkCudaErrors( hipFree( dbuf ) ) ;

	// output result on stdout
	for ( int i=0 ; NUM_THREADS>i ; i++ )
		printf( "%.4f %.4f\n", buf[2*i], buf[2*i+1] ) ;

	return EXIT_SUCCESS ;
}
#endif // P4STEREOGRAPHIC_MAIN
