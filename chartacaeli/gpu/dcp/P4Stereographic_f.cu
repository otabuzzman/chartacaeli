#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstdlib>

#include <hip/hip_runtime.h>

#include "P4Stereographic_f.h"
#include "Coordinate_f.h"
#include "Math_f.h"

// from CUDA Toolkit samples
#include <hip/hip_runtime_api.h>

__device__ P4Stereographic::P4Stereographic() {
	init( 0, 90, 1, 1 ) ;
}

__device__ void P4Stereographic::init( float lam0, float phi1, float R, float k0 ) {
	this->lam0 = lam0 ;
	this->phi1 = phi1 ;
	sincospif( phi1/180, &sinphi1, &cosphi1 ) ;
	this->R = R ;
	this->k0 = k0 ;

	if ( phi1 == 90 )
		mode = M_NORTH ;
	else if ( phi1 == -90 )
		mode = M_SOUTH ;
	else if ( phi1 == 0 )
		mode = M_EQUATOR ;
	else
		mode = M_OBLIQUE ;
}

__device__ Coordinate& P4Stereographic::forward( const Coordinate& lamphi, Coordinate& xy ) {
	float sinlamdif, coslamdif ;
	float sinphi, cosphi, k, t ;

	sincospif( ( lamphi.x-lam0 )/180, &sinlamdif, &coslamdif ) ;
	sincospif( lamphi.y/180, &sinphi, &cosphi ) ;

	switch ( mode ) {
	case M_NORTH:
		t = __tanf( radians( 45-lamphi.y/2 ) ) ;
		xy.x = 2*R*k0*t*sinlamdif ;
		xy.y = -2*R*k0*t*coslamdif ;

		break ;
	case M_SOUTH:
		t = __tanf( radians( 45+lamphi.y/2 ) ) ;
		xy.x = 2*R*k0*t*sinlamdif ;
		xy.y = 2*R*k0*t*coslamdif ;

		break ;
	case M_EQUATOR:
		k = 2*k0/( 1+cosphi*coslamdif ) ;
		xy.x = R*k*cosphi*sinlamdif ;
		xy.y = R*k*sinphi ;

		break ;
	case M_OBLIQUE:
		k = 2*k0/( 1+sinphi1*sinphi+cosphi1*cosphi*coslamdif ) ;
		xy.x = R*k*cosphi*sinlamdif ;
		xy.y = R*k*( cosphi1*sinphi-sinphi1*cosphi*coslamdif ) ;

		break ;
	}

	return xy ;
}

__device__ Coordinate& P4Stereographic::inverse( const Coordinate& xy, Coordinate& lamphi ) {
	float p, c, sinc, cosc ;

	p = sqrtf( xy.x*xy.x+xy.y*xy.y ) ;
	c = 2*degrees( atan2f( p, 2*R*k0 ) ) ;

	sincospif( c/180, &sinc, &cosc ) ;

	lamphi.y = degrees( asinf( cosc*sinphi1+( xy.y*sinc*cosphi1/p ) ) ) ;

	switch ( mode ) {
	case M_NORTH:
		lamphi.x = lam0+degrees( atan2f( xy.x, -xy.y ) ) ;

		break ;
	case M_SOUTH:
		lamphi.x = lam0+degrees( atan2f( xy.x, xy.y ) ) ;

		break ;
	case M_EQUATOR:
	case M_OBLIQUE:
		lamphi.x = lam0+degrees( atan2f( xy.x*sinc, p*cosphi1*cosc-xy.y*sinphi1*sinc ) ) ;

		break ;
	}

	return lamphi ;
}
