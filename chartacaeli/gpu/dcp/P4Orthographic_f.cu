#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstdlib>

#include "P4Orthographic.h"
#include "Coordinate.h"
#include "Math.h"

// from CUDA Toolkit samples
#include <hip/hip_runtime_api.h>

__device__ P4Orthographic::P4Orthographic() {
	init( 0, 90, 1, 1 ) ;
}

__device__ void P4Orthographic::init( float lam0, float phi1, float R, float k0 ) {
	this->lam0 = lam0 ;
	this->phi1 = phi1 ;
	sincospif( phi1/180, &sinphi1, &cosphi1 ) ;
	this->R = R ;

	if ( phi1 == 90 )
		mode = M_NORTH ;
	else if ( phi1 == -90 )
		mode = M_SOUTH ;
	else if ( phi1 == 0 )
		mode = M_EQUATOR ;
	else
		mode = M_OBLIQUE ;
}

__device__ Coordinate& P4Orthographic::forward( const Coordinate& lamphi, Coordinate& xy ) {
	float sinlamdif, coslamdif ;
	float sinphi, cosphi ;

	sincospif( ( lamphi.x-lam0 )/180, &sinlamdif, &coslamdif ) ;
	sincospif( lamphi.y/180, &sinphi, &cosphi ) ;

	xy.x = R*cosphi*sinlamdif ;

	switch ( mode ) {
	case M_NORTH:
		xy.y = -R*cosphi*coslamdif ;

		break ;
	case M_SOUTH:
		xy.y = R*cosphi*coslamdif ;

		break ;
	case M_EQUATOR:
		xy.y = R*sinphi ;

		break ;
	case M_OBLIQUE:
		xy.y = R*( cosphi1*sinphi-sinphi1*cosphi*coslamdif ) ;

		break ;
	}

	return xy ;
}

__device__ Coordinate& P4Orthographic::inverse( const Coordinate& xy, Coordinate& lamphi ) {
	float p, c, sinc, cosc ;

	p = sqrtf( xy.x*xy.x+xy.y*xy.y ) ;
	c = degrees( asinf( p/R ) ) ;

	sincospif( c/180, &sinc, &cosc ) ;

	lamphi.y = degrees( asinf( cosc*sinphi1+( xy.y*sinc*cosphi1/p ) ) ) ;

	switch ( mode ) {
	case M_NORTH:
		lamphi.x = lam0+degrees( atan2f(xy.x, -xy.y ) ) ;

		break ;
	case M_SOUTH:
		lamphi.x = lam0+degrees( atan2f(xy.x, xy.y ) ) ;

		break ;
	case M_EQUATOR:
	case M_OBLIQUE:
		lamphi.x = lam0+degrees( atan2f( xy.x*sinc, p*cosphi1*cosc-xy.y*sinphi1*sinc ) ) ;

		break ;
	}

	return lamphi ;
}
