#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstdlib>

#include <hip/hip_runtime.h>

#include "Coordinate.h"
#include "Math.h"

// from CUDA Toolkit samples
#include <hip/hip_runtime_api.h>

__device__ Coordinate::Coordinate() : x( 0 ), y( 0 ), z( 0 ) {
}

__device__ Coordinate::Coordinate( const Coordinate& c ) {
	set( c.x, c.y, c.z ) ;
}

__device__ Coordinate::Coordinate( float x, float y, float z ) {
	set( x, y, z ) ;
}

__device__ void Coordinate::set( float x, float y, float z ) {
	this->x = x ;
	this->y = y ;
	this->z = z ;
}

__device__ void Coordinate::spherical() {
	float r = sqrtf( x*x+y*y+z*z ) ;

	x = degrees( atan2f( y, x ) ) ;
	y = degrees( asinf( z/r ) ) ;
	z = 0 ;
}

__device__ void Coordinate::cartesian() {
	float sinx, cosx, siny, cosy ;

	sincospif( x/180, &sinx, &cosx ) ;
	sincospif( y/180, &siny, &cosy ) ;

	x = cosy*cosx ;
	y = cosy*sinx ;
	z = siny ;
}
