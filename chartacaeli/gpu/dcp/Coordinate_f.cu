#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstdlib>

#include <hip/hip_runtime.h>

#include "Coordinate_f.h"
#include "Math_f.h"

// from CUDA Toolkit samples
#include <hip/hip_runtime_api.h>

__device__ Coordinate::Coordinate() : x( 0 ), y( 0 ), z( 0 ) {
}

__device__ Coordinate::Coordinate( const Coordinate& c ) {
	set( c.x, c.y, c.z ) ;
}

__device__ Coordinate::Coordinate( float x, float y, float z ) {
	set( x, y, z ) ;
}

__device__ void Coordinate::set( float x, float y, float z ) {
	this->x = x ;
	this->y = y ;
	this->z = z ;
}

__device__ void Coordinate::spherical() {
	float r = sqrtf( x*x+y*y+z*z ) ;

	x = degrees( atan2f( y, x ) ) ;
	y = degrees( asinf( __fdividef( z, r ) ) ) ;
	z = 0 ;
}

__device__ void Coordinate::cartesian() {
	float sinx, cosx, siny, cosy ;

	sincospif( __fdividef( x, 180.f ), &sinx, &cosx ) ;
	sincospif( __fdividef( y, 180.f ), &siny, &cosy ) ;

	x = cosy*cosx ;
	y = cosy*sinx ;
	z = siny ;
}

#ifdef COORDINATE_MAIN
// kernel
__global__ void coordinate( float* buf ) {
	Coordinate c ;

	c.set( threadIdx.x, threadIdx.x+1, threadIdx.x+2 ) ;
	c.spherical() ;
	c.cartesian() ;
	buf[3*threadIdx.x] = c.x ;
	buf[3*threadIdx.x+1] = c.y ;
	buf[3*threadIdx.x+2] = c.z ;
}

#define NUM_BLOCKS 1
#define NUM_THREADS 360

int main( int argc, char** argv ) {
	// host buffer
	float buf[3*NUM_THREADS] ;
	// device buffer
	float* dbuf = NULL ;
	hipDeviceProp_t devProp ;
	int devID ;

	// find device and output compute capability on stderr
	devID = gpuGetMaxGflopsDeviceId() ;
	checkCudaErrors( hipSetDevice( devID ) ) ;
	checkCudaErrors( hipGetDeviceProperties( &devProp, devID ) ) ;
	fprintf( stderr, "%d%d\n", devProp.major, devProp.minor ) ;

	// allocate device buffer memory
	checkCudaErrors( hipMalloc( (void**) &dbuf, sizeof( float )*3*NUM_THREADS ) ) ;

	// run kernel
	coordinate<<<NUM_BLOCKS, NUM_THREADS>>>( dbuf ) ;

	// copy kernel results from device buffer to host
	checkCudaErrors( hipMemcpy( buf, dbuf, sizeof( float )*3*NUM_THREADS, hipMemcpyDeviceToHost ) ) ;
	checkCudaErrors( hipFree( dbuf ) ) ;

	// output result on stdout
	for ( int i=0 ; NUM_THREADS>i ; i++ )
		printf( "%.6f %.6f %.6f\n", buf[3*i], buf[3*i+1], buf[3*i+2] ) ;

	return EXIT_SUCCESS ;
}
#endif // COORDINATE_MAIN
