#include <cstdio>
#include <cstdlib>

#include <hip/hip_runtime.h>

#include "Vector4D.h"

__device__ Vector4D::Vector4D() : e0( 0 ), e1( 0 ), e2( 0 ), e3( 0 ) {
}

__device__ Vector4D::Vector4D( const Vector4D& v ) {
	set( v.e0, v.e1, v.e2, v.e3 ) ;
}

__device__ Vector4D::Vector4D( float e0, float e1, float e2, float e3 ) {
	set( e0, e1, e2, e3 ) ;
}

__device__ void Vector4D::set( float e0, float e1, float e2, float e3 ) {
	this->e0 = e0 ;
	this->e1 = e1 ;
	this->e2 = e2 ;
	this->e3 = e3 ;
}

__device__ Vector4D& Vector4D::apply( const Vector4D& matcol0, const Vector4D& matcol1, const Vector4D& matcol2, const Vector4D& matcol3 ) {
	return apply(
			matcol0.e0, matcol1.e0, matcol2.e0, matcol3.e0,
			matcol0.e1, matcol1.e1, matcol2.e1, matcol3.e1,
			matcol0.e2, matcol1.e2, matcol2.e2, matcol3.e2,
			matcol0.e3, matcol1.e3, matcol2.e3, matcol3.e3 ) ;
}

__device__ Vector4D& Vector4D::apply(
			const float m00, const float m01, const float m02, const float m03,
			const float m10, const float m11, const float m12, const float m13,
			const float m20, const float m21, const float m22, const float m23,
			const float m30, const float m31, const float m32, const float m33 ) {
	float _e0 = e0, _e1 = e1, _e2 = e2, _e3 = e3 ;

	e0 = _e0*m00+_e1*m01+_e2*m02+_e3*m03 ;
	e1 = _e0*m10+_e1*m11+_e2*m12+_e3*m13 ;
	e2 = _e0*m20+_e1*m21+_e2*m22+_e3*m23 ;
	e3 = _e0*m30+_e1*m31+_e2*m32+_e3*m33 ;

	return *this ;
}
