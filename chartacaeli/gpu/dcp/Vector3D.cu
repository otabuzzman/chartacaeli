#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstdlib>

#include <hip/hip_runtime.h>

#include "Vector3D.h"

// from CUDA Toolkit samples
#include <hip/hip_runtime_api.h>

__device__ Vector3D::Vector3D() : x( 0 ), y( 0 ), z( 0 ) {
}

__device__ Vector3D::Vector3D( double v[3] ) {
	set( v[0], v[1], v[2] ) ;
}

__device__ Vector3D::Vector3D( Vector3D& v ) {
	set( v.x, v.y, v.z ) ;
}

__device__ Vector3D::Vector3D( double x, double y, double z ) {
	set( x, y, z ) ;
}

__device__ void Vector3D::set( double x, double y, double z ) {
	this->x = x ;
	this->y = y ;
	this->z = z ;
}

__device__ Vector3D* Vector3D::add( Vector3D& v ) {
	x += v.x ;
	y += v.y ;
	z += v.z ;

	return this ;
}

__device__ Vector3D* Vector3D::sub( Vector3D& v ) {
	x -= v.x ;
	y -= v.y ;
	z -= v.z ;

	return this ;
}

__device__ Vector3D* Vector3D::mul( double a ) {
	x *= a ;
	y *= a ;
	z *= a ;

	return this ;
}

__device__ double Vector3D::dot( Vector3D& v ) {
	return x*v.x+y*v.y+z*v.z ;
}

__device__ Vector3D* Vector3D::cross( Vector3D& v ) {
	double x, y, z ;

	x = this->x ;
	y = this->y ;
	z = this->z ;

	this->x = y*v.z-z*v.y ;
	this->y = z*v.x-x*v.z ;
	this->z = x*v.y-y*v.x ;

	return this ;
}

__device__ double* Vector3D::toArray() {
	double* r = new double[3] ;

	r[0] = x ;
	r[1] = y ;
	r[2] = z ;

	return r ;
}

#ifdef VECTOR3D_MAIN
// kernel
__global__ void vector3d( double* buf ) {
	Vector3D *a, *b ;
	int i = threadIdx.x ;

	a = new Vector3D( i, i+.123, i+.234 ) ;
	b = new Vector3D( i+.234, i+.123, i ) ;
	buf[i] = a->cross( *b )->dot( *a ) ;

	delete b ;
	delete a ;
}

#define NUM_BLOCKS 1
#define NUM_THREADS 360

int main( int argc, char** argv ) {
	// host buffer
	double buf[NUM_THREADS] ;
	// device buffer
	double* dbuf = NULL ;
	hipDeviceProp_t devProp ;
	int devID ;

	// find device and output compute capability on stderr
	devID = gpuGetMaxGflopsDeviceId() ;
	checkCudaErrors( hipSetDevice( devID ) ) ;
	checkCudaErrors( hipGetDeviceProperties( &devProp, devID ) ) ;
	fprintf( stderr, "%d%d\n", devProp.major, devProp.minor ) ;

	// allocate device buffer memory
	checkCudaErrors( hipMalloc( (void**) &dbuf, sizeof( double )*NUM_THREADS ) ) ;

	// run kernel
	vector3d<<<NUM_BLOCKS, NUM_THREADS>>>( dbuf ) ;

	// copy kernel results from device buffer to host
	checkCudaErrors( hipMemcpy( buf, dbuf, sizeof( double )*NUM_THREADS, hipMemcpyDeviceToHost ) ) ;
	checkCudaErrors( hipFree( dbuf ) ) ;

	// output result on stdout
	for ( int i=0 ; NUM_THREADS>i ; i++ )
		printf( "%.6f\n", buf[i] ) ;

	return EXIT_SUCCESS ;
}
#endif // VECTOR3D_MAIN
