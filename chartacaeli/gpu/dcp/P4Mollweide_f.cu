#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstdlib>

#include <hip/hip_math_constants.h>

#include "P4Mollweide_f.h"
#include "Coordinate_f.h"
#include "Math_f.h"

// from CUDA Toolkit samples
#include <hip/hip_runtime_api.h>

__device__ P4Mollweide::P4Mollweide() : V_CON( 1e-7f ),
								radperdeg( 3.141592653589793f/180.f ) ,
								degperrad( 180.f/3.141592653589793f ) {
	init( 0, 0, 1, 1 ) ;
}

__device__ void P4Mollweide::init( float lam0, float phi1, float R, float k0 ) {
	this->lam0 = lam0 ;
	this->R = R ;
}

__device__ Coordinate& P4Mollweide::forward( const Coordinate& lamphi, Coordinate& xy ) {
	float tht2 = lamphi.y, dtht2 = 0, sintht2, costht2 ;
	float sinphi, tht, sintht, costht ;

	sinphi = sinpif( lamphi.y/180 ) ;

	do {
		tht2 = tht2+dtht2 ;

		sincospif( tht2/180, &sintht2, &costht2 ) ;

		dtht2 = -( tht2*radperdeg+sintht2-HIP_PI_F*sinphi )/( 1+costht2 )*degperrad ;
	} while ( abs( dtht2 )>V_CON ) ;

	tht = tht2*.5f ;
	sincospif( tht/180, &sintht, &costht ) ;

	xy.x = ( 2.82842712475f/HIP_PI_F )*R*( lamphi.x-lam0 )*costht*radperdeg ;
	xy.y = 1.41421356237f*R*sintht ;

	return xy ;
}

__device__ Coordinate& P4Mollweide::inverse( const Coordinate& xy, Coordinate& lamphi ) {
	float tht, sin2tht, costht ;

	tht = degrees( asinf( xy.y/( 1.41421356237f*R ) ) ) ;

	sin2tht = sinpif( ( 2*tht )/180 ) ;
	lamphi.y = degrees( asinf( ( 2*tht*radperdeg+sin2tht )/HIP_PI_F ) ) ;

	if ( abs( lamphi.y ) == 90 )
		lamphi.x = lam0 ;
	else {
		costht = cospif( tht/180 ) ;
		lamphi.x = lam0+( HIP_PI_F*xy.x/( 2.82842712475f*R*costht ) )*degperrad ;
	}

	return lamphi ;
}
