#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstdlib>

#include <hip/hip_math_constants.h>

#include "P4Mollweide_f.h"
#include "Coordinate_f.h"
#include "Math_f.h"

// from CUDA Toolkit samples
#include <hip/hip_runtime_api.h>

__device__ P4Mollweide::P4Mollweide() {
	init( 0, 0, 1, 1 ) ;
}

__device__ void P4Mollweide::init( float lam0, float phi1, float R, float k0 ) {
	this->lam0 = lam0 ;
	this->R = R ;
}

__device__ Coordinate& P4Mollweide::forward( const Coordinate& lamphi, Coordinate& xy ) {
	float tht2 = lamphi.y, dtht2 = 0, sintht2, costht2 ;
	float sinphi, tht, sintht, costht ;

	sinphi = sinpif( __fdividef( lamphi.y, 180.f ) ) ;

	do {
		tht2 = tht2+dtht2 ;

		sincospif( __fdividef( tht2, 180.f ), &sintht2, &costht2 ) ;

		dtht2 = __fdividef( -( tht2*radperdeg+sintht2-HIP_PI_F*sinphi ), ( 1+costht2 )*degperrad ) ;
	} while ( abs( dtht2 )>V_CON ) ;

	tht = tht2*.5f ;
	sincospif( __fdividef( tht, 180.f ), &sintht, &costht ) ;

	xy.x = ( __fdividef( 2.82842712475f, HIP_PI_F ) )*R*( lamphi.x-lam0 )*costht*radperdeg ;
	xy.y = 1.41421356237f*R*sintht ;

	return xy ;
}

__device__ Coordinate& P4Mollweide::inverse( const Coordinate& xy, Coordinate& lamphi ) {
	float tht, sin2tht, costht ;

	tht = degrees( asinf( __fdividef( xy.y, ( 1.41421356237f*R ) ) ) ) ;

	sin2tht = sinpif( __fdividef( ( 2*tht ), 180.f ) ) ;
	lamphi.y = degrees( asinf( __fdividef( ( 2*tht*radperdeg+sin2tht ), HIP_PI_F ) ) ) ;

	if ( abs( lamphi.y ) == 90 )
		lamphi.x = lam0 ;
	else {
		costht = cospif( __fdividef( tht, 180.f ) ) ;
		lamphi.x = lam0+( HIP_PI_F*__fdividef( xy.x, ( 2.82842712475f*R*costht ) ) )*degperrad ;
	}

	return lamphi ;
}

#ifdef P4MOLLWEIDE_MAIN
// kernel
__global__ void p4mollweide( float* buf ) {
	P4Mollweide proj ;
	Coordinate lamphi, xy, res ;
	int i = threadIdx.x ;

	lamphi.set( (float) i, (float) ( i%90 ), 0 ) ;
	proj.forward( lamphi, xy ) ;
	proj.inverse( xy, res ) ;
	buf[2*i] = res.x ;
	buf[2*i+1] = res.y ;
}

#define NUM_BLOCKS 1
#define NUM_THREADS 360

int main( int argc, char** argv ) {
	// host buffer
	float buf[2*NUM_THREADS] ;
	// device buffer
	float* dbuf = NULL ;
	hipDeviceProp_t devProp ;
	int devID ;

	// find device and output compute capability on stderr
	devID = gpuGetMaxGflopsDeviceId() ;
	checkCudaErrors( hipSetDevice( devID ) ) ;
	checkCudaErrors( hipGetDeviceProperties( &devProp, devID ) ) ;
	fprintf( stderr, "%d%d\n", devProp.major, devProp.minor ) ;

	// allocate device buffer memory
	checkCudaErrors( hipMalloc( (void**) &dbuf, sizeof( float )*2*NUM_THREADS ) ) ;

	// run kernel
	p4mollweide<<<NUM_BLOCKS, NUM_THREADS>>>( dbuf ) ;

	// copy kernel results from device buffer to host
	checkCudaErrors( hipMemcpy( buf, dbuf, sizeof( float )*2*NUM_THREADS, hipMemcpyDeviceToHost ) ) ;
	checkCudaErrors( hipFree( dbuf ) ) ;

	// output result on stdout
	for ( int i=0 ; NUM_THREADS>i ; i++ )
		printf( "%.4f %.4f\n", buf[2*i], buf[2*i+1] ) ;

	return EXIT_SUCCESS ;
}
#endif // P4MOLLWEIDE_MAIN
