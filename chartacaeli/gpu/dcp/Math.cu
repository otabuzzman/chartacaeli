#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstdlib>

#include <hip/hip_runtime.h>

// from CUDA Toolkit samples
#include <hip/hip_runtime_api.h>

// names inspired by Cg Toolkit Standard Library
__device__ double degrees( double a ) {
	return a*0.017453292519943295769236907685 ;
}
__device__ double radians( double a ) {
	return a*57.295779513082320876798154814105 ;
}

// kernel
__global__ void kernel( double* buf ) {
	buf[threadIdx.x] = degrees( ( sin( radians( (double) threadIdx.x ) ) ) ) ;
}

#define NUM_BLOCKS 1
#define NUM_THREADS 360


int main( int argc, char** argv ) {
	findCudaDevice( argc, (const char**) argv ) ;

	double buf[NUM_THREADS] ;
	double* dbuf = NULL ;
	checkCudaErrors( hipMalloc( (void**) &dbuf, sizeof( double )*NUM_THREADS ) ) ;

	kernel<<<NUM_BLOCKS, NUM_THREADS>>>( dbuf ) ;

	checkCudaErrors( hipMemcpy( buf, dbuf, sizeof( double )*NUM_THREADS, hipMemcpyDeviceToHost ) ) ;
	checkCudaErrors( hipFree( dbuf ) ) ;

	for ( int i=0 ; NUM_THREADS>i ; i++ )
		printf( "%.8f\n", buf[i] ) ;

	return EXIT_SUCCESS ;
}
