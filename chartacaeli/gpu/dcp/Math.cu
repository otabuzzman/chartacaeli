#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstdlib>

#include <hip/hip_runtime.h>

// from CUDA Toolkit samples
#include <hip/hip_runtime_api.h>

// names inspired by Cg Toolkit Standard Library
__device__ double degrees( double a ) {
	return a*57.295779513082320876798154814105 ;
}
__device__ double radians( double a ) {
	return a*0.017453292519943295769236907685 ;
}

// kernel
__global__ void kernel( double* buf ) {
	buf[threadIdx.x] = degrees( asin ( sin( radians( (double) threadIdx.x ) ) ) ) ;
}

#define NUM_BLOCKS 1
#define NUM_THREADS 360


int main( int argc, char** argv ) {
	// host buffer
	double buf[NUM_THREADS] ;
	// device buffer
	double* dbuf = NULL ;
	hipDeviceProp_t devProp ;
	int devID ;

	// find device and output compute capability on stderr
	devID = gpuGetMaxGflopsDeviceId() ;
	checkCudaErrors( hipSetDevice( devID ) ) ;
	checkCudaErrors( hipGetDeviceProperties( &devProp, devID ) ) ;
	fprintf( stderr, "%d%d\n", devProp.major, devProp.minor ) ;

	// allocate device buffer memory
	checkCudaErrors( hipMalloc( (void**) &dbuf, sizeof( double )*NUM_THREADS ) ) ;

	// run kernel
	kernel<<<NUM_BLOCKS, NUM_THREADS>>>( dbuf ) ;

	// copy kernel results from device buffer to host
	checkCudaErrors( hipMemcpy( buf, dbuf, sizeof( double )*NUM_THREADS, hipMemcpyDeviceToHost ) ) ;
	checkCudaErrors( hipFree( dbuf ) ) ;

	// output result on stdout
	for ( int i=0 ; NUM_THREADS>i ; i++ )
		printf( "%.8f\n", buf[i] ) ;

	return EXIT_SUCCESS ;
}
