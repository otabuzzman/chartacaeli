#include <cstdio>
#include <cstdlib>

#include <hip/hip_runtime.h>

#include "Vector3D.h"

// from CUDA Toolkit samples
#include <hip/hip_runtime_api.h>

__device__ Vector3D::Vector3D() : Coordinate( 0, 0, 0 ) {
}

__device__ Vector3D::Vector3D( const Vector3D& v ) : Coordinate( v.x, v.y, v.z ) {
}

__device__ Vector3D::Vector3D( float x, float y, float z ) : Coordinate( x, y, z ) {
}

__device__ Vector3D& Vector3D::add( const Vector3D& v ) {
	x += v.x ;
	y += v.y ;
	z += v.z ;

	return *this ;
}

__device__ Vector3D& Vector3D::sub( const Vector3D& v ) {
	x -= v.x ;
	y -= v.y ;
	z -= v.z ;

	return *this ;
}

__device__ Vector3D& Vector3D::mul( float a ) {
	x *= a ;
	y *= a ;
	z *= a ;

	return *this ;
}

__device__ float Vector3D::dot( const Vector3D& v ) {
	return x*v.x+y*v.y+z*v.z ;
}

__device__ Vector3D& Vector3D::cross( const Vector3D& v ) {
	float x, y, z ;

	x = this->x ;
	y = this->y ;
	z = this->z ;

	this->x = y*v.z-z*v.y ;
	this->y = z*v.x-x*v.z ;
	this->z = x*v.y-y*v.x ;

	return *this ;
}

__device__ Vector3D& Vector3D::apply( const Vector3D& matcol0, const Vector3D& matcol1, const Vector3D& matcol2 ) {
	return apply(
			matcol0.x, matcol1.x, matcol2.x,
			matcol0.y, matcol1.y, matcol2.y,
			matcol0.z, matcol1.z, matcol2.z ) ;
}

__device__ Vector3D& Vector3D::apply(
			const float m00, const float m01, const float m02,
			const float m10, const float m11, const float m12,
			const float m20, const float m21, const float m22 ) {
	float _x = x, _y = y, _z = z ;

	x = _x*m00+_y*m01+_z*m02 ;
	y = _x*m10+_y*m11+_z*m12 ;
	z = _x*m20+_y*m21+_z*m22 ;

	return *this ;
}
