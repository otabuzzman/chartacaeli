#include "hip/hip_runtime.h"
#include <cstdlib>

#include "dcp/P4Projector.h"
#include "dcp/P4Stereographic.h"
#include "dcp/P4Orthographic.h"
#include "dcp/P4Mollweide.h"
#include "dcp/RealMatrix.h"
#include "dcp/Vector3D.h"
#include "dcp/Plane.h"
#include "dcp/Coordinate.h"

__device__ P4Projector* createP4Projector( const char *pnam ) {
	switch ( *( pnam+2 ) ) {
		case 'S':
			return new P4Stereographic() ;
		case 'O':
			return new P4Orthographic() ;
		case 'M':
			return new P4Mollweide() ;
		default:
			return NULL ;
	}
}

// CUDA kernel
extern "C" __global__ void run(
			const char* pnam, const double lam0, const double phi1, const double R, const double k0,
			const double m2p00, const double m2p01, const double m2p02,
			const double m2p10, const double m2p11, const double m2p12,
			const double m2p20, const double m2p21, const double m2p22,
			const double h2t00, const double h2t01, const double h2t02, const double h2t03,
			const double h2t10, const double h2t11, const double h2t12, const double h2t13,
			const double h2t20, const double h2t21, const double h2t22, const double h2t23,
			const double h2t30, const double h2t31, const double h2t32, const double h2t33,
			const double p1x, const double p1y, const double p1z,
			const double p2x, const double p2y, const double p2z,
			const double p3x, const double p3y, const double p3z,
			const int dimo, const int dimp, const int** texture,
			const int dims, const int dimt, int** mapping,
			const double ups ) {
	int t, s ;
	P4Projector* proj ;
	RealMatrix m2p(
		m2p00, m2p01, m2p02,
		m2p10, m2p11, m2p12,
		m2p20, m2p21, m2p22 ) ;
	RealMatrix h2t(
		h2t00, h2t01, h2t02, h2t03,
		h2t10, h2t11, h2t12, h2t13,
		h2t20, h2t21, h2t22, h2t23,
		h2t30, h2t31, h2t32, h2t33 ) ;
	Plane spt( p1x, p1y, p1z, p2x, p2y, p2z, p3x, p3y, p3z ) ;
	double st[] = { 0, 0, 1 }, *t0, *op, ca[] = { 0, 0, 0, 1 } ;
	Coordinate uv, *eq ;
	Vector3D l0, l1, *t1 ;

	t = blockIdx.y*blockDim.y+threadIdx.y ;
	s = blockIdx.x*blockDim.x+threadIdx.x ;

	if ( t>=dimt || s>=dims )
		return ;

	proj = createP4Projector( pnam ) ;
	proj->init( lam0, phi1, R, k0 ) ;

	st[1] = t*ups ;
	st[0] = s*ups ;

	// transform s/t to projection coordinates u/v
	t0 = m2p.operate( st ) ;
	uv.set( t0[0], t0[1], t0[2] ) ;
	// transform u/v to spherical (equatorial) coordinates
	eq = proj->inverse( uv ) ;
	// convert spherical to cartesian
	eq->cartesian() ;
	l1.set( eq->x, eq->y, eq->z ) ;
	// find cartesian coordinates c/a of spatial intersection with texture
	t1 = spt.intersection( l0, l1 ) ;
	ca[0] = t1->x ;
	ca[1] = t1->y ;
	ca[2] = t1->z ;
	// transform c/a to texture coordinates o/p
	op = h2t.operate( ca ) ;

	// map o/p if on texture
	if ( op[0]>=0 && op[1]>=0 && dimo>op[0] && dimp>op[1] )
		mapping[t][s] = texture[(int) op[1]][(int) op[0]] ;

	delete[] op ;
	delete t1 ;
	delete eq ;
	delete[] t0 ;
	delete proj ;
}

#ifdef PJ2TEXTUREMAPPERGPU_MAIN
int main( int argc, char** argv ) {
	return EXIT_SUCCESS ;
}
#endif // PJ2TEXTUREMAPPERGPU_MAIN
