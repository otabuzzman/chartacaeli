
#include <hip/hip_runtime.h>
// projector params in global memory
__device__ double lim0 ;
__device__ double phi1 ;
__device__ double R ;
__device__ double k0 ;

// texture params (dimo, dimp) in global memory
__device__ double dimo ;
__device__ double dimp ;

// mapping params (dims, dimt) in global memory
__device__ double dims ;
__device__ double dimt ;

// general params in global memory
__device__ double ups ;

// devive kernel
extern "C" __global__ void run(
	const char* pnam,
	const double** tmM2P,
	const double** tmH2T,
	const double** spT,
	const int** texture,
	const int** mapping ) {
}
