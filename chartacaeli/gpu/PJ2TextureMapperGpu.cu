#include "hip/hip_runtime.h"
#include <cstdlib>

#include "dcp/P4Projector.h"
#include "dcp/P4Stereographic.h"
#include "dcp/P4Orthographic.h"
#include "dcp/P4Mollweide.h"
#include "dcp/RealMatrix.h"
#include "dcp/Vector3D.h"
#include "dcp/Plane.h"
#include "dcp/Coordinate.h"

__device__ P4Projector* createP4Projector( const char *pnam ) {
	switch ( *( pnam+2 ) ) {
		case 'S':
			return new P4Stereographic() ;
		case 'O':
			return new P4Orthographic() ;
		case 'M':
			return new P4Mollweide() ;
		default:
			return NULL ;
	}
}

// projector params in global memory
__device__ double lim0 ;
__device__ double phi1 ;
__device__ double R ;
__device__ double k0 ;

// texture params (dimo, dimp) in global memory
__device__ int dimo ;
__device__ int dimp ;

// mapping params (dims, dimt) in global memory
__device__ int dims ;
__device__ int dimt ;

// general params in global memory
__device__ double ups ;

// CUDA kernel
extern "C" __global__ void run( const char* pnam, double* tmM2P, double* tmH2T, double** spT, const int** texture, int** mapping ) {
	int t, s ;
	P4Projector* proj ;
	RealMatrix m2p( tmM2P, 3, 3 ), h2t( tmH2T, 4, 4 ) ;
	Vector3D p1( spT[0][0], spT[0][1], spT[0][2] ) ;
	Vector3D p2( spT[1][0], spT[1][1], spT[1][2] ) ;
	Vector3D p3( spT[2][0], spT[2][1], spT[2][2] ) ;
	Plane spt( p1, p2, p3 ) ;
	double st[] = { 0, 0, 1 }, *t0, *op, ca[] = { 0, 0, 0, 1 } ;
	Coordinate uv, *eq ;
	Vector3D l0, l1, *t1 ;

	t = blockIdx.y*blockDim.y+threadIdx.y ;
	s = blockIdx.x*blockDim.x+threadIdx.x ;

	if ( t>=dimt || s>=dims )
		return ;

	proj = createP4Projector( pnam ) ;
	proj->init( lim0, phi1, R, k0 ) ;

	st[1] = t*ups ;
	st[0] = s*ups ;

	// transform s/t to projection coordinates u/v
	t0 = m2p.operate( st ) ;
	uv.set( t0[0], t0[1], t0[2] ) ;
	// transform u/v to spherical (equatorial) coordinates
	eq = proj->inverse( uv ) ;
	// convert spherical to cartesian
	eq->cartesian() ;
	l1.set( eq->x, eq->y, eq->z ) ;
	// find cartesian coordinates c/a of spatial intersection with texture
	t1 = spt.intersection( l0, l1 ) ;
	ca[0] = t1->x ;
	ca[1] = t1->y ;
	ca[2] = t1->z ;
	// transform c/a to texture coordinates o/p
	op = h2t.operate( ca ) ;

	// map o/p if on texture
	if ( op[0]>=0 && op[1]>=0 && dimo>op[0] && dimp>op[1] )
		mapping[t][s] = texture[(int) op[1]][(int) op[0]] ;

	delete[] op ;
	delete t1 ;
	delete[] t0 ;
	delete proj ;
}

#ifdef PJ2TEXTUREMAPPERGPU_MAIN
int main( int argc, char** argv ) {
	return EXIT_SUCCESS ;
}
#endif // PJ2TEXTUREMAPPERGPU_MAIN
