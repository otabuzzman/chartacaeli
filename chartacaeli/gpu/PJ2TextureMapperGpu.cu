#include "hip/hip_runtime.h"
#include <cstdlib>

#include "dcp/P4Projector.h"
#include "dcp/P4Stereographic.h"
#include "dcp/P4Orthographic.h"
#include "dcp/P4Mollweide.h"
#include "dcp/RealMatrix.h"
#include "dcp/Vector3D.h"
#include "dcp/Plane.h"
#include "dcp/Coordinate.h"

__device__ P4Projector* createP4Projector( const char *pnam ) {
	switch ( *( pnam+2 ) ) {
		case 'S':
			return new P4Stereographic() ;
		case 'O':
			return new P4Orthographic() ;
		case 'M':
			return new P4Mollweide() ;
		default:
			return NULL ;
	}
}

// projector params in global memory
__device__ double lim0 ;
__device__ double phi1 ;
__device__ double R ;
__device__ double k0 ;

// texture params (dimo, dimp) in global memory
__device__ int dimo ;
__device__ int dimp ;

// mapping params (dims, dimt) in global memory
__device__ int dims ;
__device__ int dimt ;

// general params in global memory
__device__ double ups ;

// CUDA kernel
extern "C" __global__ void run( const char* pnam, double* tmM2Pj, double* tmH2Tj, double** spTj, const int** texture, int** mapping ) {
	int t, s ;
	P4Projector* proj ;
	RealMatrix *tmM2P, *tmH2T ;
	Vector3D *p1, *p2, *p3 ;
	Plane* spT ;
	double st[] = { 0, 0, 1 }, *t0, *op, ca[] = { 0, 0, 0, 1 } ;
	Coordinate uv, *eq, *t1 ;
	Vector3D l0, l1, *t2 ;

	t = blockIdx.y*blockDim.y+threadIdx.y ;
	s = blockIdx.x*blockDim.x+threadIdx.x ;

	if ( dimt>t || dims>s )
		return ;

	proj = createP4Projector( pnam ) ;
	proj->init( lim0, phi1, R, k0 ) ;

	tmM2P = new RealMatrix( tmM2Pj, 3, 3 ) ;
	tmH2T = new RealMatrix( tmH2Tj, 4, 4 ) ;

	p1 = new Vector3D( spTj[0] ) ;
	p2 = new Vector3D( spTj[1] ) ;
	p3 = new Vector3D( spTj[2] ) ;
	spT = new Plane( *p1, *p2, *p3 ) ;

	st[1] = t*ups ;
	st[0] = s*ups ;

	// transform s/t to projection coordinates u/v
	t0 = tmM2P->operate( st ) ;
	uv.set( t0[0], t0[1], t0[2] ) ;
	// transform u/v to spherical (equatorial) coordinates
	eq = proj->inverse( uv ) ;
	// convert spherical to cartesian
	t1 = eq->cartesian() ;
	l1.set( t1->x, t1->y, t1->z ) ;
	// find cartesian coordinates c/a of spatial intersection with texture
	t2 = spT->intersection( l0, l1 ) ;
	ca[0] = t2->x ;
	ca[1] = t2->y ;
	ca[2] = t2->z ;
	// transform c/a to texture coordinates o/p
	op = tmH2T->operate( ca ) ;

	// map o/p if on texture
	if ( op[0]>=0 && op[1]>=0 && dimo>op[0] && dimp>op[1] )
		mapping[t][s] = texture[(int) op[1]][(int) op[0]] ;

	delete op ;
	delete t2 ;
	delete t1 ;
	delete eq ;
	delete t0 ;
	delete spT ;
	delete p3 ;
	delete p2;
	delete p1 ;
	delete tmH2T ;
	delete tmM2P ;
	delete proj ;
}

#ifdef PJ2TEXTUREMAPPERGPU_MAIN
int main( int argc, char** argv ) {
	return EXIT_SUCCESS ;
}
#endif // PJ2TEXTUREMAPPERGPU_MAIN
