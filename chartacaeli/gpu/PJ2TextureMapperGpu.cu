#include "hip/hip_runtime.h"
#include <new>
#include <cstdlib>

#include "dcp/P4Projector.h"
#include "dcp/P4Stereographic.h"
#include "dcp/P4Orthographic.h"
#include "dcp/P4Mollweide.h"
#include "dcp/Plane.h"
#include "dcp/Vector4D.h"
#include "dcp/Vector3D.h"
#include "dcp/Coordinate.h"

__device__ P4Projector* createP4Projector( const char *pnam, unsigned char* pmem ) {
	switch ( *( pnam+2 ) ) {
		case 'S':
			return new( pmem ) P4Stereographic() ;
		case 'O':
			return new( pmem ) P4Orthographic() ;
		case 'M':
			return new( pmem ) P4Mollweide() ;
		default:
			return nullptr ;
	}
}

// CUDA kernel
extern "C" __global__ void run(
			const char* pnam, const double lam0, const double phi1, const double R, const double k0,
			const double m2p00, const double m2p01, const double m2p02,
			const double m2p10, const double m2p11, const double m2p12,
			const double m2p20, const double m2p21, const double m2p22,
			const double h2t00, const double h2t01, const double h2t02, const double h2t03,
			const double h2t10, const double h2t11, const double h2t12, const double h2t13,
			const double h2t20, const double h2t21, const double h2t22, const double h2t23,
			const double h2t30, const double h2t31, const double h2t32, const double h2t33,
			const double p1x, const double p1y, const double p1z,
			const double p2x, const double p2y, const double p2z,
			const double p3x, const double p3y, const double p3z,
			const int dimo, const int dimp, const int** texture,
			const int dims, const int dimt, int** mapping,
			const double ups ) {
	int t, s ;
	__shared__ unsigned char pool[256] ;
	__shared__ P4Projector* proj ;
	__shared__ Plane* spt ;
	Vector3D uv, l0, l1, ca ;
	Vector4D op ;

	t = blockIdx.y*blockDim.y+threadIdx.y ;
	s = blockIdx.x*blockDim.x+threadIdx.x ;

	if ( t>=dimt || s>=dims )
		return ;

	if ( threadIdx.y == 0 && threadIdx.x == 0 ) {
		proj = createP4Projector( pnam, &pool[0] ) ;
		if ( proj == nullptr )
			return ;
		proj->init( lam0, phi1, R, k0 ) ;

		spt = new( &pool[128] ) Plane( p1x, p1y, p1z, p2x, p2y, p2z, p3x, p3y, p3z ) ;
	}

	__syncthreads() ;

	// transform s/t to projection coordinates u/v
	uv.set( s*ups, t*ups, 1 ) ;
	uv.apply(
		m2p00, m2p01, m2p02,
		m2p10, m2p11, m2p12,
		m2p20, m2p21, m2p22 ) ;

	// transform u/v to spherical (equatorial) coordinates
	proj->inverse( uv, l1 ) ;

	// convert spherical to cartesian
	l1.cartesian() ;

	// find spatial intersection with texture
	spt->intersection( l0, l1, ca ) ;

	// transform to texture coordinates o/p
	op.set( ca.x, ca.y, ca.z, 1 ) ;
	op.apply(
		h2t00, h2t01, h2t02, h2t03,
		h2t10, h2t11, h2t12, h2t13,
		h2t20, h2t21, h2t22, h2t23,
		h2t30, h2t31, h2t32, h2t33 ) ;

	// map o/p if on texture
	if ( op.e0>=0 && op.e1>=0 && dimo>op.e0 && dimp>op.e1 )
		mapping[t][s] = texture[(int) op.e1][(int) op.e0] ;
}

#ifdef PJ2TEXTUREMAPPERGPU_MAIN
int main( int argc, char** argv ) {
	return EXIT_SUCCESS ;
}
#endif // PJ2TEXTUREMAPPERGPU_MAIN
