#include "hip/hip_runtime.h"
#include <cstdlib>

#include "dcp/P4Projector.h"
#include "dcp/P4Stereographic.h"
#include "dcp/P4Orthographic.h"
#include "dcp/P4Mollweide.h"

__device__ P4Projector* createP4Projector( const char *pnam ) {
	P4Projector* proj ;

	switch ( *( pnam+2 ) ) {
		case 'S':
			return new P4Stereographic() ;
		case 'O':
			return new P4Orthographic() ;
		case 'M':
			return new P4Mollweide() ;
		default:
			break ;
	}

	return NULL ;
}

// projector params in global memory
__device__ double lim0 ;
__device__ double phi1 ;
__device__ double R ;
__device__ double k0 ;

// texture params (dimo, dimp) in global memory
__device__ double dimo ;
__device__ double dimp ;

// mapping params (dims, dimt) in global memory
__device__ double dims ;
__device__ double dimt ;

// general params in global memory
__device__ double ups ;

// CUDA kernel
extern "C" __global__ void run(
	const char* pnam,
	const double** tmM2P,
	const double** tmH2T,
	const double** spT,
	const int** texture,
	const int** mapping ) {
}

int main( int argc, char** argv ) {
	return EXIT_SUCCESS ;
}
