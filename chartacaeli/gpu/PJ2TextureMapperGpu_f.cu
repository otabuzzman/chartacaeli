#include "hip/hip_runtime.h"
#include <new>

#include "dcp/P4Projector_f.h"
#include "dcp/P4Stereographic_f.h"
#include "dcp/P4Orthographic_f.h"
#include "dcp/P4Mollweide_f.h"
#include "dcp/Plane_f.h"
#include "dcp/Vector4D_f.h"
#include "dcp/Vector3D_f.h"
#include "dcp/Coordinate_f.h"

// CUDA kernel
extern "C" __global__ void run(
			const char* pnam, const float lam0, const float phi1, const float R, const float k0,
			const float m2p00, const float m2p01, const float m2p02,
			const float m2p10, const float m2p11, const float m2p12,
			const float m2p20, const float m2p21, const float m2p22,
			const float h2t00, const float h2t01, const float h2t02, const float h2t03,
			const float h2t10, const float h2t11, const float h2t12, const float h2t13,
			const float h2t20, const float h2t21, const float h2t22, const float h2t23,
			const float h2t30, const float h2t31, const float h2t32, const float h2t33,
			const float p1x, const float p1y, const float p1z,
			const float p2x, const float p2y, const float p2z,
			const float p3x, const float p3y, const float p3z,
			const int dimo, const int dimp, const int** texture,
			const int dims, const int dimt, int** mapping,
			const float ups ) {
	int t, s, p, o ;
	__shared__ unsigned char pool[256] ;
	__shared__ P4Projector* proj ;
	__shared__ Plane* spt ;
	Vector3D uv, l0, l1, ca ;
	Vector4D op ;

	t = blockIdx.y*blockDim.y+threadIdx.y ;
	s = blockIdx.x*blockDim.x+threadIdx.x ;

	if ( t>=dimt || s>=dims )
		return ;

	if ( threadIdx.y == 0 && threadIdx.x == 0 ) {
		switch ( *( pnam+2 ) ) {
			case 'S':
				proj = new( &pool[0] ) P4Stereographic() ;
				break ;
			case 'O':
				proj = new( &pool[0] ) P4Orthographic() ;
				break ;
			case 'M':
				proj = new( &pool[0] ) P4Mollweide() ;
				break ;
			default:
				asm( "trap ;" ) ;
		}

		spt = new( &pool[128] ) Plane( p1x, p1y, p1z, p2x, p2y, p2z, p3x, p3y, p3z ) ;
	}

	__syncthreads() ;

	// transform s/t to projection coordinates u/v
	uv.set( s*ups, t*ups, 1 ) ;
	uv.apply(
		m2p00, m2p01, m2p02,
		m2p10, m2p11, m2p12,
		m2p20, m2p21, m2p22 ) ;

	// transform u/v to spherical (equatorial) coordinates
	proj->init( lam0, phi1, R, k0 ) ;
	proj->inverse( uv, l1 ) ;

	// convert spherical to cartesian
	l1.cartesian() ;

	// find spatial intersection with texture
	spt->intersection( l0, l1, ca ) ;

	// transform to texture coordinates o/p
	op.set( ca.x, ca.y, ca.z, 1 ) ;
	op.apply(
		h2t00, h2t01, h2t02, h2t03,
		h2t10, h2t11, h2t12, h2t13,
		h2t20, h2t21, h2t22, h2t23,
		h2t30, h2t31, h2t32, h2t33 ) ;

	// map o/p if on texture
	p = (int) op.e1 ;
	o = (int) op.e0 ;
	if ( (unsigned int) p<dimp && (unsigned int) o<dimo )
		mapping[t][s] = texture[p][o] ;
}

#ifdef PJ2TEXTUREMAPPERGPU_MAIN
#include <hip/hip_runtime.h>

// from CUDA Toolkit samples
#include <hip/hip_runtime_api.h>

int main( int argc, char** argv ) {
	hipDeviceProp_t devProp ;
	int devID ;
	unsigned char buf[3] ;
	const char *h_pnam = "P4Stereographic" ;
	char *d_pnam ;
	float lam0, phi1, R, k0 ;
	float m2p00, m2p01, m2p02 ;
	float m2p10, m2p11, m2p12 ;
	float m2p20, m2p21, m2p22 ;
	float h2t00, h2t01, h2t02, h2t03 ;
	float h2t10, h2t11, h2t12, h2t13 ;
	float h2t20, h2t21, h2t22, h2t23 ;
	float h2t30, h2t31, h2t32, h2t33 ;
	float p1x, p1y, p1z ;
	float p2x, p2y, p2z ;
	float p3x, p3y, p3z ;
	int dimo, dimp, *texture, **h_texture, **d_texture ;
	int dims, dimt, *mapping, **h_mapping, **d_mapping ;
	float ups ;

	lam0 = 0.0f ; phi1 = 90.0f ; R = 118.8f ; k0 = 1.0f ;
	m2p00 = 0.5175793085449578f ;  m2p01 = 0.8556352373331308f ; m2p02 = -27.61474066215803f ;
	m2p10 = -0.8556352373331311f ; m2p11 = 0.5175793085449577f ; m2p12 = -100.85847174398357f ;
	m2p20 = 0.0f ;                 m2p21 = 0.0f ;                m2p22 = 1.0f ;
	h2t00 = -3402.8858901402996f ;    h2t01 = -1220.9974680683858f ;   h2t02 = -3419.719730243767f ;      h2t03 = 5148.3305361527955f ;
	h2t10 = -4519.169481016445f ;     h2t11 = -933.648235895421f ;     h2t12 = -2788.9911878218313f ;     h2t13 = 5525.831502438334f ;
	h2t20 = 19.06000692773117f ;      h2t21 = 7.920202046020817f ;     h2t22 = 13.632963600546473f ;      h2t23 = -24.275149004126146f ;
	h2t30 = -1.289164294604532E-14f ; h2t31 = -5.10584599744509E-15f ; h2t32 = -1.0503453106267556E-14f ; h2t33 = 1.000000000000018f ;
	p1x = 0.7360690204252228f ; p1y = 0.16255538665963573f ; p1z = 0.6570982753273586f ;
	p2x = 0.8458053524679503f ; p2y = 0.14657741169931554f ; p2z = 0.5129603962452617f ;
	p3x = 0.6338551355901809f ; p3y = 0.3804772484154413f ;  p3z = 0.6733978991088165f ;
	dimo = 512 ; dimp = 512 ;
	dims = 1360 ; dimt = 1517 ;
	ups = 0.06349999259166753f ;

	// find device and output compute capability on stderr
	devID = gpuGetMaxGflopsDeviceId() ;
	checkCudaErrors( hipSetDevice( devID ) ) ;
	checkCudaErrors( hipGetDeviceProperties( &devProp, devID ) ) ;
	fprintf( stderr, "%d%d\n", devProp.major, devProp.minor ) ;

	// allocate device memeory for projector name
	checkCudaErrors( hipMalloc( (void**) &d_pnam, strlen( h_pnam )+1 ) ) ;
	checkCudaErrors( hipMemcpy( d_pnam, h_pnam, strlen( h_pnam )+1, hipMemcpyHostToDevice ) ) ;

	// allocate host memory for texture
	texture = (int*) malloc( dimo*dimp*sizeof( int ) ) ;
	// initialize texture with RGB data
	for ( int i=0 ; dimo*dimp>i ; i++ ) {
		fread( &buf[0], 1, 3, stdin ) ;
		if ( ferror( stdin ) || feof( stdin ) )
			break ;
		texture[i] = buf[0]<<16|buf[1]<<8|buf[2] ;
	}
	// allocate device memory pendant and copy texture from host (mind 'array of arrays' type)
	h_texture = (int**) malloc( dimp*sizeof( int* ) ) ;
	for ( int i=0 ; dimp>i ; i++ ) {
		checkCudaErrors( hipMalloc( (void**) &h_texture[i], dimo*sizeof( int ) ) ) ;
		checkCudaErrors( hipMemcpy( h_texture[i], &texture[i*dimo], dimo*sizeof( int ), hipMemcpyHostToDevice ) ) ;
	}
	checkCudaErrors( hipMalloc( (void**) &d_texture, dimp*sizeof( int* ) ) ) ;
	checkCudaErrors( hipMemcpy( d_texture, h_texture, dimp*sizeof( int* ), hipMemcpyHostToDevice ) ) ;

	// allocate host...
	mapping = (int*) malloc( dims*dimt*sizeof( int ) ) ;
	// ...and device memory for mapping (mind 'array of arrays' type)
	h_mapping = (int**) malloc( dimt*sizeof( int* ) ) ;
	for ( int i=0 ; dimt>i ; i++ )
		checkCudaErrors( hipMalloc( (void**) &h_mapping[i], dims*sizeof( int ) ) ) ;
	checkCudaErrors( hipMalloc( (void**) &d_mapping, dimt*sizeof( int* ) ) ) ;
	checkCudaErrors( hipMemcpy( d_mapping, h_mapping, dimt*sizeof( int* ), hipMemcpyHostToDevice ) ) ;

	// run kernel
	dim3 gdim( ( dims+32-1 )/32, ( dimt+32-1 )/32 ) ;
	dim3 bdim( 32, 32 ) ;
	run<<<gdim, bdim>>>(
			d_pnam, lam0, phi1, R, k0,
			m2p00, m2p01, m2p02,
			m2p10, m2p11, m2p12,
			m2p20, m2p21, m2p22,
			h2t00, h2t01, h2t02, h2t03,
			h2t10, h2t11, h2t12, h2t13,
			h2t20, h2t21, h2t22, h2t23,
			h2t30, h2t31, h2t32, h2t33,
			p1x, p1y, p1z,
			p2x, p2y, p2z,
			p3x, p3y, p3z,
			dimo, dimp, (const int**) d_texture,
			dims, dimt, d_mapping,
			ups ) ;

	// copy mapping from device back to host
	for ( int i=0 ; dimt>i ; i++ )
		checkCudaErrors( hipMemcpy( &mapping[i*dims], h_mapping[i], dims*sizeof( int ), hipMemcpyDeviceToHost ) ) ;
	// output mapping result RGB data
	for ( int i=0 ; dims*dimt>i ; i++ ) {
		buf[0] = mapping[i]>>16&255 ;
		buf[1] = mapping[i]>>8&255 ;
		buf[2] = mapping[i]&255 ;
		fwrite( &buf[0], 1, 3, stdout ) ;
	}

	for ( int i=0 ; dimt>i ; i++ )
		checkCudaErrors( hipFree( h_mapping[i] ) ) ;
	checkCudaErrors( hipFree( d_mapping ) ) ;
	free( h_mapping ) ;
	free( mapping ) ;

	for ( int i=0 ; dimp>i ; i++ )
		checkCudaErrors( hipFree( h_texture[i] ) ) ;
	checkCudaErrors( hipFree( d_texture ) ) ;
	free( h_texture ) ;
	free( texture ) ;

	checkCudaErrors( hipFree( d_pnam ) ) ;

	return EXIT_SUCCESS ;
}
#endif // PJ2TEXTUREMAPPERGPU_MAIN



// runtime values from tf-0032
//	lam0 = 0.0f ;
//	phi1 = 90.0f ;
//	R = 118.8f ;
//	k0 = 1.0f ;
//	m2p00 = 0.5175793085449578f ;
//	m2p01 = 0.8556352373331308f ;
//	m2p02 = -27.61474066215803f ;
//	m2p10 = -0.8556352373331311f ;
//	m2p11 = 0.5175793085449577f ;
//	m2p12 = -100.85847174398357f ;
//	m2p20 = 0.0f ;
//	m2p21 = 0.0f ;
//	m2p22 = 1.0f ;
//	h2t00 = -3402.8858901402996f ;
//	h2t01 = -1220.9974680683858f ;
//	h2t02 = -3419.719730243767f ;
//	h2t03 = 5148.3305361527955f ;
//	h2t10 = -4519.169481016445f ;
//	h2t11 = -933.648235895421f ;
//	h2t12 = -2788.9911878218313f ;
//	h2t13 = 5525.831502438334f ;
//	h2t20 = 19.06000692773117f ;
//	h2t21 = 7.920202046020817f ;
//	h2t22 = 13.632963600546473f ;
//	h2t23 = -24.275149004126146f ;
//	h2t30 = -1.289164294604532E-14f ;
//	h2t31 = -5.10584599744509E-15f ;
//	h2t32 = -1.0503453106267556E-14f ;
//	h2t33 = 1.000000000000018f ;
//	p1x = 0.7360690204252228f ;
//	p1y = 0.16255538665963573f ;
//	p1z = 0.6570982753273586f ;
//	p2x = 0.8458053524679503f ;
//	p2y = 0.14657741169931554f ;
//	p2z = 0.5129603962452617f ;
//	p3x = 0.6338551355901809f ;
//	p3y = 0.3804772484154413f ;
//	p3z = 0.6733978991088165f ;
//	dimo = 512 ;
//	dimp = 512 ;
//	dims = 1360 ;
//	dimt = 1517 ;
//	ups = 0.06349999259166753f ;
